#include "hip/hip_runtime.h"
#include "cuda_current_device.hpp"
#include "cuda_utils_common.hpp"
#include "cuda_utils.hpp" // for fix_debug workaround

hipDevice_t get_cu_device()
{
    hipDevice_t device_id;
    int cuda_device_id;
    hipGetDevice( &cuda_device_id );
    CU_SAFE_CALL(hipDeviceGet(&device_id, cuda_device_id));
    return device_id;
}

// -----------------------------------------------------------------------------

bool check_kernel_size(hipDevice_t device_id, const int3 block_size, const int3 grid_size)
{
    int3 max_block_size;
    int3 max_grid_size;
    CU_SAFE_CALL(hipDeviceGetAttribute(&max_block_size.x, hipDeviceAttributeMaxBlockDimX, device_id) );
    CU_SAFE_CALL(hipDeviceGetAttribute(&max_block_size.y, hipDeviceAttributeMaxBlockDimY, device_id) );
    CU_SAFE_CALL(hipDeviceGetAttribute(&max_block_size.z, hipDeviceAttributeMaxBlockDimZ, device_id) );

    CU_SAFE_CALL(hipDeviceGetAttribute(&max_grid_size.x, hipDeviceAttributeMaxGridDimX, device_id) );
    CU_SAFE_CALL(hipDeviceGetAttribute(&max_grid_size.y, hipDeviceAttributeMaxGridDimY, device_id) );
    CU_SAFE_CALL(hipDeviceGetAttribute(&max_grid_size.z, hipDeviceAttributeMaxGridDimZ, device_id) );

    return  (grid_size.x  <= max_grid_size.x ) && (grid_size.y  <= max_grid_size.y ) && (grid_size.z  <= max_grid_size.z) &&
            (block_size.x <= max_block_size.x) && (block_size.y <= max_block_size.y) && (block_size.z <= max_block_size.z);
}

// -----------------------------------------------------------------------------

bool check_kernel_size(hipDevice_t device, int block_size, int grid_size)
{
    if(block_size < 0 || grid_size < 0) return false;
    return check_kernel_size(device, make_int3(block_size, 0, 0), make_int3(grid_size, 0, 0));
}
