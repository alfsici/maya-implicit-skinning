#include "hip/hip_runtime.h"
/*
 Implicit skinning
 Copyright (C) 2013 Rodolphe Vaillant, Loic Barthe, Florian Cannezin,
 Gael Guennebaud, Marie Paule Cani, Damien Rohmer, Brian Wyvill,
 Olivier Gourmel

 This program is free software: you can redistribute it and/or modify
 it under the terms of the GNU General Public License 3 as published by
 the Free Software Foundation.

 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.

 You should have received a copy of the GNU General Public License
 along with this program. If not, see <http://www.gnu.org/licenses/>
 */
#include "cuda_current_device.hpp"

#include "cuda_utils_common.hpp"

// -----------------------------------------------------------------------------

hipDevice_t get_cu_device()
{
    hipDevice_t device_id;
    int cuda_device_id;
    hipGetDevice( &cuda_device_id );
    CU_SAFE_CALL(hipDeviceGet(&device_id, cuda_device_id));
    return device_id;
}

// -----------------------------------------------------------------------------

bool check_kernel_size(hipDevice_t device_id, const int3 block_size, const int3 grid_size)
{
    int3 max_block_size;
    int3 max_grid_size;
    CU_SAFE_CALL(hipDeviceGetAttribute(&max_block_size.x, hipDeviceAttributeMaxBlockDimX, device_id) );
    CU_SAFE_CALL(hipDeviceGetAttribute(&max_block_size.y, hipDeviceAttributeMaxBlockDimY, device_id) );
    CU_SAFE_CALL(hipDeviceGetAttribute(&max_block_size.z, hipDeviceAttributeMaxBlockDimZ, device_id) );

    CU_SAFE_CALL(hipDeviceGetAttribute(&max_grid_size.x, hipDeviceAttributeMaxGridDimX, device_id) );
    CU_SAFE_CALL(hipDeviceGetAttribute(&max_grid_size.y, hipDeviceAttributeMaxGridDimY, device_id) );
    CU_SAFE_CALL(hipDeviceGetAttribute(&max_grid_size.z, hipDeviceAttributeMaxGridDimZ, device_id) );

    return  (grid_size.x  <= max_grid_size.x ) && (grid_size.y  <= max_grid_size.y ) && (grid_size.z  <= max_grid_size.z) &&
            (block_size.x <= max_block_size.x) && (block_size.y <= max_block_size.y) && (block_size.z <= max_block_size.z);
}

// -----------------------------------------------------------------------------

bool check_kernel_size(hipDevice_t device, int block_size, int grid_size)
{
    if(block_size < 0 || grid_size < 0) return false;
    return check_kernel_size(device, make_int3(block_size, 0, 0), make_int3(grid_size, 0, 0));
}

// -----------------------------------------------------------------------------
