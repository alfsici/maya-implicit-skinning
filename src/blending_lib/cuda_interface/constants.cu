/*
 Implicit skinning
 Copyright (C) 2013 Rodolphe Vaillant, Loic Barthe, Florian Cannezin,
 Gael Guennebaud, Marie Paule Cani, Damien Rohmer, Brian Wyvill,
 Olivier Gourmel

 This program is free software: you can redistribute it and/or modify
 it under the terms of the GNU General Public License 3 as published by
 the Free Software Foundation.

 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.

 You should have received a copy of the GNU General Public License
 along with this program. If not, see <http://www.gnu.org/licenses/>
 */
//#include "textures.hpp"
#include "cuda_utils.hpp"
#include "vec3_cu.hpp"
#include "constants.hpp"

// =============================================================================
namespace Constants{
// =============================================================================

float* d_constants;
const int nb_constants = NB_CONST;
float h_constants[nb_constants];
bool binded = false;

texture<float, 1,  hipReadModeElementType> constants_tex;

// -----------------------------------------------------------------------------

void allocate(){
    int size = nb_constants * sizeof(float);
    CUDA_SAFE_CALL(hipMalloc((void**)(&d_constants),size));
}

void free(){
    hipFree(d_constants);
    d_constants = 0;
}

// -----------------------------------------------------------------------------

extern void bind();
extern void unbind();

// -----------------------------------------------------------------------------

void update(){
    unbind();
    int size = nb_constants * sizeof(float);
    CUDA_SAFE_CALL(hipMemcpy(d_constants,
                              h_constants,
                              size,
                              hipMemcpyHostToDevice));
    bind();
}

// -----------------------------------------------------------------------------

void  set(int var, float value){ h_constants[var] = value; }
float get(int var)             { return h_constants[var];  }

// -----------------------------------------------------------------------------

void incr(int var, float value, float minv, float maxv){
    h_constants[var] += value;
    if(h_constants[var] > maxv)
        h_constants[var] = maxv;
    if(h_constants[var] < minv)
        h_constants[var] = minv;
}

// -----------------------------------------------------------------------------

void init(){
    set(B0, -1.f);
    set(B1, 0.f);
    set(B2, 1.f);
    set(F0, 1.f);
    set(F1, 0.f);
    set(F2, 0.995f);
    set(POW0, 1.f);
    set(POW1, 1.f);

    set(K0, 1.f);
    set(K1, 1.f);
    set(K2, 1.f);
    set(K3, 1.f);
    set(K4, 1.f);
    set(K5, 1.f);
}

// -----------------------------------------------------------------------------

};
// END CONSTANTS NAMESPACE =====================================================
