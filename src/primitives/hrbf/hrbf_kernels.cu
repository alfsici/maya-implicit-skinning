#include "hip/hip_runtime.h"
#include "hrbf_kernels.hpp"
#include "hrbf_env.hpp"

#include <iostream>

// =============================================================================
namespace HRBF_kernels{
// =============================================================================

__global__
void hrbf_transform_ker(const int nb_verts,
                        const float4* in_vertices,
                        const float4* in_alpha_beta,
                        const int* map_transfos,
                        const Transfo* transfos,
                        float4* out_vertices,
                        float4* out_alpha_beta)
{
    const int p = blockIdx.x * blockDim.x + threadIdx.x;

    if(p < nb_verts)
    {
        const int tr_index = map_transfos[p];
        const Transfo tr = transfos[tr_index];

        const float4  tmp    = in_alpha_beta[p];
        const Vec3_cu beta   = Vec3_cu(tmp.x, tmp.y, tmp.z);
        const Vec3_cu beta_t = tr * beta;

        out_alpha_beta[p] = make_float4(beta_t.x,beta_t.y,beta_t.z,tmp.w);

        const float4   tmp2      = in_vertices[p];
        const Point_cu point     = Point_cu(tmp2.x, tmp2.y, tmp2.z);
        const Point_cu point_t   = tr * point;

        out_vertices[p] = make_float4(point_t.x, point_t.y, point_t.z, tmp.w);
    }
}

// -----------------------------------------------------------------------------

/// Transform each vertex of each rbf primitive
/// @param d_transform Map for a bone parent index its rigid transformation
/// (tab[parent[ith_bone]] = ith_bone_transformation)
void hrbf_transform(const Cuda_utils::Device::Array<Transfo>& d_transform,
                    const Cuda_utils::DA_int& d_map_transfos)
{
    if(HRBF_env::d_init_points.size() == 0) return;

    const int block_size = 16;
    const int grid_size  =
            (HRBF_env::d_init_points.size() + block_size - 1) / block_size;

    HRBF_env::unbind();

    hrbf_transform_ker
            <<<grid_size, block_size >>>
            (HRBF_env::d_init_points.size(),
             HRBF_env::d_init_points.ptr(),
             HRBF_env::d_init_alpha_beta.ptr(),
             d_map_transfos.ptr(),
             d_transform.ptr(),
             HRBF_env::hd_points.d_ptr(),
             HRBF_env::hd_alphas_betas.d_ptr());

    HRBF_env::hd_points.update_host_mem();
    HRBF_env::hd_alphas_betas.update_host_mem();

    CUDA_CHECK_ERRORS();

    HRBF_env::bind();
}

}// END HRBF_ENV NAMESPACE =====================================================
