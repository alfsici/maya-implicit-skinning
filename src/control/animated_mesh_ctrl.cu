#include "hip/hip_runtime.h"
/*
 Implicit skinning
 Copyright (C) 2013 Rodolphe Vaillant, Loic Barthe, Florian Cannezin,
 Gael Guennebaud, Marie Paule Cani, Damien Rohmer, Brian Wyvill,
 Olivier Gourmel

 This program is free software: you can redistribute it and/or modify
 it under the terms of the GNU General Public License 3 as published by
 the Free Software Foundation.

 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.

 You should have received a copy of the GNU General Public License
 along with this program. If not, see <http://www.gnu.org/licenses/>
 */
#include "animated_mesh_ctrl.hpp"

#include "animesh.hpp"
#include "std_utils.hpp"
#include "skeleton.hpp"
#include "timer.hpp"
#include "precomputed_prim.hpp"
#include "hrbf_env.hpp"

// -----------------------------------------------------------------------------
namespace { __device__ void fix_debug() { } }

// This takes ownership of the Mesh (and gives it to Animesh).  It does not take ownership
// of Skeleton.
Animated_mesh_ctrl::Animated_mesh_ctrl(const Mesh *mesh, std::shared_ptr<const Skeleton> skel_) :
    _nb_iter(7),
    skel(skel_),
    _animesh(new Animesh(mesh, skel_))
{
}

Animated_mesh_ctrl::~Animated_mesh_ctrl()
{
    delete _animesh;
}

void Animated_mesh_ctrl::set_do_smoothing(bool state)
{
    if(_animesh != 0){
        _animesh->set_smooth_mesh(state);
    }
}

void Animated_mesh_ctrl::set_smooth_factor(int i, float fact){
    _animesh->set_smooth_factor(i, fact);
}

void Animated_mesh_ctrl::update_base_potential()
{
    assert(_animesh != 0);
    _animesh->update_base_potential();
}

void Animated_mesh_ctrl::get_base_potential(std::vector<float> &pot, std::vector<Vec3_cu> &grad) const  { assert(_animesh != NULL); _animesh->get_base_potential(pot, grad); }
void Animated_mesh_ctrl::set_base_potential(const std::vector<float> &pot, const std::vector<Vec3_cu> &grad) { assert(_animesh != NULL); _animesh->set_base_potential(pot, grad); }

// -----------------------------------------------------------------------------

void Animated_mesh_ctrl::deform_mesh()
{
    _animesh->transform_vertices();
}

void Animated_mesh_ctrl::read_weights(std::ifstream& file,
                                      std::vector<float4>& weights )
{
    std::string nil;
    int nb_samples = -1;
    file >> nil /*'nb_points'*/ >> nb_samples;
    weights.resize( nb_samples );

    for(int j = 0; j < nb_samples; ++j)
    {
        Vec3_cu beta;
        float alpha;
        file >> beta.x >> beta.y >> beta.z;
        file >> alpha;

        weights[j] = make_float4(beta.x, beta.y, beta.z, alpha);
    }
}

void Animated_mesh_ctrl::read_hrbf_env_weights(
        std::ifstream& file,
        std::vector<std::vector<float4> >& bone_weights)
{
    std::string nil;
    int bone_id  = -1;
    int nb_bones_file = 0;

    file >> nil/*'nb_bone'*/ >> nb_bones_file;

    for(int i = 0; i < nb_bones_file; i++ )
    {
        file >> nil/*'bone_id'*/ >> bone_id;
        read_weights(file, bone_weights[i]);
    }
}

void Animated_mesh_ctrl::load_ism(const char* filename)
{
    using namespace std;
    ifstream file(filename);
/*
    if(!file.is_open()) {
        cerr << "Error importing file " << filename << endl;
        return;
    }

    std::vector<float> radius_hrbf(_animesh->get_skel()->nb_joints(), -1.f      );

    std::vector<std::vector<float4> > bone_weights(_animesh->get_skel()->nb_joints());

    while( !file.eof() )
    {
        string section;
        file >> section;

        if(section == "[HRBF_ENV_WEIGHTS]") read_hrbf_env_weights(file, bone_weights);
        else
        {
            std::cerr << "WARNING ism import: can't read this symbol '";
            std::cerr << section << "'" << std::endl;
        }
    }
    file.close();

    _animesh->update_base_potential();
    */
}

void Animated_mesh_ctrl::set_nb_iter_smooting(int nb_iter)
{
    if(_animesh != 0){
        _animesh->set_smoothing_iter(nb_iter);
        _nb_iter = nb_iter;
    }
}

void Animated_mesh_ctrl::smooth_conservative(){
    _animesh->set_smoothing_type(EAnimesh::CONSERVATIVE);
}
void Animated_mesh_ctrl::smooth_laplacian(){
    _animesh->set_smoothing_type(EAnimesh::LAPLACIAN);
}
void Animated_mesh_ctrl::smooth_tangential(){
    _animesh->set_smoothing_type(EAnimesh::TANGENTIAL);
}
void Animated_mesh_ctrl::smooth_humphrey(){
    _animesh->set_smoothing_type(EAnimesh::HUMPHREY);
}
void Animated_mesh_ctrl::set_local_smoothing(bool state){
    _animesh->set_local_smoothing(state);
}
void Animated_mesh_ctrl::set_smooth_force_a (float alpha){
    _animesh->set_smooth_force_a(alpha);
}
void Animated_mesh_ctrl::set_smooth_force_b (float beta){
    _animesh->set_smooth_force_b(beta);
}

void Animated_mesh_ctrl::set_smooth_smear(float val ){
    _animesh->set_smooth_smear(val);
}

void Animated_mesh_ctrl::set_smoothing_weights_diffusion_iter(int nb_iter){
    _animesh->set_smoothing_weights_diffusion_iter(nb_iter);
}

void Animated_mesh_ctrl::get_anim_vertices_aifo(std::vector<Point_cu>& out) const
{
    _animesh->get_anim_vertices_aifo(out);
}

void Animated_mesh_ctrl::copy_vertices(const std::vector<Vec3_cu> &vertices)
{
    _animesh->copy_vertices(vertices);
}

int Animated_mesh_ctrl::get_nb_vertices() const
{
    return _animesh->get_nb_vertices();
}
