#include "hip/hip_runtime.h"
/*
 Implicit skinning
 Copyright (C) 2013 Rodolphe Vaillant, Loic Barthe, Florian Cannezin,
 Gael Guennebaud, Marie Paule Cani, Damien Rohmer, Brian Wyvill,
 Olivier Gourmel

 This program is free software: you can redistribute it and/or modify
 it under the terms of the GNU General Public License 3 as published by
 the Free Software Foundation.

 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.

 You should have received a copy of the GNU General Public License
 along with this program. If not, see <http://www.gnu.org/licenses/>
 */
#include "animesh.hpp"

#include "animesh_kers.hpp"
#include "macros.hpp"
#include "vec3_cu.hpp"
#include "distance_field.hpp"
#include "conversions.hpp"
#include "std_utils.hpp"
#include "skeleton.hpp"

// -----------------------------------------------------------------------------

#include <fstream>
#include <sstream>
#include <cstring>
#include <limits>
#include <cmath>

using namespace Cuda_utils;

namespace { __device__ void fix_debug() { } }

Animesh::Animesh(const Mesh *m_, const Skeleton* s_) :
    _mesh(m_), _skel(s_),
    mesh_smoothing(EAnimesh::LAPLACIAN),
    do_smooth_mesh(false),
    do_local_smoothing(true),
    smoothing_iter(7),
    diffuse_smooth_weights_iter(6),
    smooth_force_a(0.5f),
    smooth_force_b(0.5f),
    smooth_smear(0.f),
    d_input_smooth_factors(_mesh->get_nb_vertices()),
    d_smooth_factors_conservative(_mesh->get_nb_vertices(), 0.f),
    d_smooth_factors_laplacian(_mesh->get_nb_vertices()),
    d_input_vertices(_mesh->get_nb_vertices()),
    d_edge_lengths(_mesh->get_nb_edges()),
    d_edge_mvc(_mesh->get_nb_edges()),
    d_vertices_state(_mesh->get_nb_vertices()),
    d_vertices_states_color(EAnimesh::NB_CASES),
//    d_input_normals(m->get_nb_vertices()),
    d_output_vertices(_mesh->get_nb_vertices()),
    d_gradient(_mesh->get_nb_vertices()),
    d_input_tri(_mesh->get_nb_tri()*3),
    d_edge_list(_mesh->get_nb_edges()),
    d_edge_list_offsets(2 * _mesh->get_nb_vertices()),
    d_base_potential(_mesh->get_nb_vertices()),
    d_base_gradient(_mesh->get_nb_vertices()),
    d_piv(_mesh->get_nb_faces()),
    d_unpacked_normals(_mesh->get_nb_vertices() * _mesh->_max_faces_per_vertex),
    d_unpacked_tangents(_mesh->get_nb_vertices() * _mesh->_max_faces_per_vertex),
    h_vert_buffer(_mesh->get_nb_vertices()),
    d_vert_buffer(_mesh->get_nb_vertices()),
    d_vert_buffer_2(_mesh->get_nb_vertices()),
    d_vals_buffer(_mesh->get_nb_vertices())
{

    int nb_vert = _mesh->get_nb_vertices();
    Host::Array<EAnimesh::Vert_state> h_vert_state(nb_vert, EAnimesh::NOT_DISPLACED);

    d_vertices_state.copy_from(h_vert_state);

    d_vertices_states_color.set(EAnimesh::POTENTIAL_PIT      , make_float4(1.f, 0.f, 1.f, 0.99f)); // purple
    d_vertices_states_color.set(EAnimesh::GRADIENT_DIVERGENCE, make_float4(1.f, 0.f, 0.f, 0.99f)); // red
    d_vertices_states_color.set(EAnimesh::NB_ITER_MAX        , make_float4(0.f, 0.f, 1.f, 0.99f)); // blue
    d_vertices_states_color.set(EAnimesh::NOT_DISPLACED      , make_float4(1.f, 1.f, 0.f, 0.99f)); // yellow
    d_vertices_states_color.set(EAnimesh::FITTED             , make_float4(0.f, 1.f, 0.f, 0.99f)); // green
    d_vertices_states_color.set(EAnimesh::OUT_VERT           , make_float4(1.f, 1.f, 1.f, 0.99f)); // white
    d_vertices_states_color.set(EAnimesh::NORM_GRAD_NULL     , make_float4(0.f, 0.f, 0.f, 0.99f)); // black

    // Not mandatory but it is supposed to accelerate a little bit animation
    // when activated

    // Fill the attributes in device memory
    copy_mesh_data(*_mesh);

    init_smooth_factors(d_input_smooth_factors);
    init_vert_to_fit();

    compute_mvc();

    update_base_potential();
}

// -----------------------------------------------------------------------------

Animesh::~Animesh()
{
}

// -----------------------------------------------------------------------------

void Animesh::init_vert_to_fit()
{
    int nb_vert = _mesh->get_nb_vertices();
    std::vector<int> h_vert_to_fit_base;
    h_vert_to_fit_base.reserve(nb_vert);
    int acc = 0;
    for (int i = 0; i < nb_vert; ++i)
    {
        if( !_mesh->is_disconnect(i) ){
            h_vert_to_fit_base.push_back( i );
            acc++;
        }
    }

    d_vert_to_fit.     malloc(acc);
    d_vert_to_fit_base.malloc(acc);

    d_vert_to_fit_buff_scan.malloc(acc+1);
    d_vert_to_fit_buff.malloc(acc);
    h_vert_to_fit_buff.malloc(acc);

    d_vert_to_fit_base.copy_from(h_vert_to_fit_base);
    d_vert_to_fit.     copy_from(h_vert_to_fit_base);
}

// -----------------------------------------------------------------------------

void Animesh::copy_vertices(const std::vector<Vec3_cu> &vertices)
{
    assert(vertices.size() == d_input_vertices.size());
    const int nb_vert = vertices.size();
    Host::Array<Point_cu > input_vertices(nb_vert);

    for(int i = 0; i < nb_vert; i++)
        input_vertices[i] = Convs::to_point(vertices[i]);

    d_input_vertices.copy_from(input_vertices);
}

void Animesh::copy_mesh_data(const Mesh& a_mesh)
{
    const int nb_vert = a_mesh.get_nb_vertices();

    Host::Array<Point_cu > input_vertices(nb_vert);
    for(int i = 0; i < nb_vert; i++)
    {
        Point_cu  pos = Convs::to_point( a_mesh.get_vertex(i) );
        input_vertices[i] = pos;
    }

    int n_faces = a_mesh.get_nb_faces();
    Host::Array<Mesh::PrimIdxVertices> h_piv(n_faces);
    for(int i = 0; i < n_faces; i++){
        h_piv[i] = a_mesh.get_piv(i);
    }
    d_piv.copy_from(h_piv);

    d_input_vertices.copy_from(input_vertices);

    HA_int h_edge_list(a_mesh.get_nb_edges());
    HA_int h_edge_list_offsets(2*nb_vert);
    for(int i = 0; i < a_mesh.get_nb_edges(); i++){
        h_edge_list[i] = a_mesh.get_edge(i);
    }
    for(int i = 0; i < nb_vert; i++){
        h_edge_list_offsets[2*i  ] = a_mesh.get_edge_offset(2*i  );
        h_edge_list_offsets[2*i+1] = a_mesh.get_edge_offset(2*i+1);
    }
    d_edge_list.copy_from(h_edge_list);
    d_edge_list_offsets.copy_from(h_edge_list_offsets);

    Cuda_utils::mem_cpy_htd(d_input_tri. ptr(), a_mesh.get_tri_index(), a_mesh.get_nb_tri()*3 );
}

// -----------------------------------------------------------------------------



void Animesh::compute_mvc()
{
    //Device::Array<Vec3_cu> d_grad( d_input_vertices.size() );
    Host::Array<float> edge_lengths(_mesh->get_nb_edges());
    Host::Array<float> edge_mvc    (_mesh->get_nb_edges());
    for(int i = 0; i < _mesh->get_nb_vertices(); i++)
    {
        Point_cu pos = Convs::to_point( _mesh->get_vertex(i)      );
        Vec3_cu  nor = Convs::to_point( _mesh->get_mean_normal(i) ); // FIXME : should be the gradient

        Mat3_cu frame = Mat3_cu::coordinate_system( nor ).transpose();
        float sum = 0.f;
        bool  out = false;
        // Look up neighborhood
        int dep      = _mesh->get_edge_offset(i*2    );
        int nb_neigh = _mesh->get_edge_offset(i*2 + 1);
        int end      = (dep+nb_neigh);

        if( nor.norm() < 0.00001f || _mesh->is_vert_on_side(i) ) {
            for(int n = dep; n < end; n++) edge_mvc[n] = 0.f;
        }
        else
        {
            for(int n = dep; n < end; n++)
            {
                int id_curr = _mesh->get_edge( n );
                int id_next = _mesh->get_edge( (n+1) >= end  ? dep   : n+1 );
                int id_prev = _mesh->get_edge( (n-1) <  dep  ? end-1 : n-1 );

                // compute edge length
                Point_cu  curr = Convs::to_point( _mesh->get_vertex(id_curr) );
                Vec3_cu e_curr = (curr - pos);
                edge_lengths[n] = e_curr.norm();

                // compute mean value coordinates
                // coordinates are computed by projecting the neighborhood to the
                // tangent plane
                {
                    // Project on tangent plane
                    Vec3_cu e_next = Convs::to_point( _mesh->get_vertex(id_next) ) - pos;
                    Vec3_cu e_prev = Convs::to_point( _mesh->get_vertex(id_prev) ) - pos;

                    e_curr = frame * e_curr;
                    e_next = frame * e_next;
                    e_prev = frame * e_prev;

                    e_curr.x = 0.f;
                    e_next.x = 0.f;
                    e_prev.x = 0.f;

                    float norm_curr_2D = e_curr.norm();

                    e_curr.normalize();
                    e_next.normalize();
                    e_prev.normalize();

                    // Computing mvc
                    float anext = std::atan2( -e_prev.z * e_curr.y + e_prev.y * e_curr.z, e_prev.dot(e_curr) );
                    float aprev = std::atan2( -e_curr.z * e_next.y + e_curr.y * e_next.z, e_curr.dot(e_next) );

                    float mvc = 0.f;
                    if(norm_curr_2D > 0.0001f)
                        mvc = (std::tan(anext*0.5f) + std::tan(aprev*0.5f)) / norm_curr_2D;

                    sum += mvc;
                    edge_mvc[n] = mvc;
                    out = out || mvc < 0.f;
                }
            }
            // we ignore points outside the convex hull
            if( sum  <= 0.f || out || isnan(sum) ) {
                for(int n = dep; n < end; n++) edge_mvc[n] = 0.f;
            }
        }

    }
    d_edge_lengths.copy_from( edge_lengths );
    d_edge_mvc.    copy_from( edge_mvc     );
}

void Animesh::init_smooth_factors(Cuda_utils::DA_float& d_smooth_factors)
{
    const int nb_vert = _mesh->get_nb_vertices();
    HA_float smooth_factors(nb_vert);

    for(int i=0; i<nb_vert; i++)
        smooth_factors[i] = 0.0f;

    d_smooth_factors.copy_from(smooth_factors);
}

// -----------------------------------------------------------------------------

void Animesh::diffuse_attr(int nb_iter, float strength, float *attr)
{
    Animesh_kers::diffuse_values(attr,
                            d_vals_buffer.ptr(),
                            d_edge_list,
                            d_edge_list_offsets,
                            strength,
                            nb_iter);
}

// -----------------------------------------------------------------------------

void Animesh::get_anim_vertices_aifo(std::vector<Point_cu>& anim_vert)
{
    const int nb_vert = d_output_vertices.size();
    Cuda_utils::HA_Point_cu h_out_verts(nb_vert);
    h_out_verts.copy_from(d_output_vertices);

    anim_vert.insert(anim_vert.end(), &h_out_verts[0], &h_out_verts[0] + nb_vert);
}

#include "cuda_utils_thrust.hpp"

__global__ static
void transform_vert_to_fit(const int* src, int* dst, const int nb_vert)
{
    const int p = blockIdx.x * blockDim.x + threadIdx.x;
    if(p < nb_vert) dst[p] = src[p] < 0 ? 0 : 1;
}

/// here src must be different from dst
__global__ static
void pack(const int* prefix_sum, const int* src, int* dst, const int nb_vert)
{
    const int p = blockIdx.x * blockDim.x + threadIdx.x;
    if(p < nb_vert){
        const int elt = src[p];
        if(elt >= 0) dst[ prefix_sum[p] ] = elt;
    }
}

int Animesh::pack_vert_to_fit_gpu(
        Cuda_utils::Device::Array<int>& d_vert_to_fit,
        Cuda_utils::Device::Array<int>& buff,
        Cuda_utils::Device::Array<int>& packed_array,
        int nb_vert_to_fit)
{
    if(nb_vert_to_fit == 0) return 0;
    assert(d_vert_to_fit.size() >= nb_vert_to_fit           );
    assert(buff.size()          >= d_vert_to_fit.size() + 1 );
    assert(packed_array.size()  >= d_vert_to_fit.size()     );

    const int block_s = 16;
    const int grid_s  = (nb_vert_to_fit + block_s - 1) / block_s;
    transform_vert_to_fit<<<grid_s, block_s >>>(d_vert_to_fit.ptr(), buff.ptr()+1, nb_vert_to_fit);
    buff.set(0, 0);// First element to zero
    CUDA_CHECK_ERRORS();

    // Compute prefix sum in buff between [1 nb_vert_to_fit]
    Cuda_utils::inclusive_scan(0, nb_vert_to_fit-1, buff.ptr()+1);

    const int new_nb_vert_to_fit = buff.fetch(nb_vert_to_fit);

    pack<<<grid_s, block_s >>>(buff.ptr(), d_vert_to_fit.ptr(), packed_array.ptr(), nb_vert_to_fit);
    CUDA_CHECK_ERRORS();

    //Cuda_utils::mem_cpy_dtd(d_vert_to_fit.ptr(), packed_array.ptr(), new_nb_vert_to_fit);

    return new_nb_vert_to_fit;
}
