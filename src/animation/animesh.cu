#include "hip/hip_runtime.h"
/*
 Implicit skinning
 Copyright (C) 2013 Rodolphe Vaillant, Loic Barthe, Florian Cannezin,
 Gael Guennebaud, Marie Paule Cani, Damien Rohmer, Brian Wyvill,
 Olivier Gourmel

 This program is free software: you can redistribute it and/or modify
 it under the terms of the GNU General Public License 3 as published by
 the Free Software Foundation.

 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.

 You should have received a copy of the GNU General Public License
 along with this program. If not, see <http://www.gnu.org/licenses/>
 */
#include "animesh.hpp"

#include "animesh_kers.hpp"
#include "macros.hpp"
#include "vec3_cu.hpp"
#include "distance_field.hpp"
#include "conversions.hpp"
#include "std_utils.hpp"
#include "skeleton.hpp"

// -----------------------------------------------------------------------------

#include <fstream>
#include <sstream>
#include <cstring>
#include <limits>
#include <cmath>

using namespace Cuda_utils;

// -----------------------------------------------------------------------------

float distsqToSeg(const Point_cu& v, const Point_cu& p1, const Point_cu& p2);

// -----------------------------------------------------------------------------

Animesh::Animesh(Mesh* m_, Skeleton* s_) :
    _mesh(m_), _skel(s_),
    mesh_smoothing(EAnimesh::LAPLACIAN),
    do_implicit_skinning(false),
    do_smooth_mesh(false),
    do_local_smoothing(true),
    do_interleave_fitting(false),
    do_update_potential(true),
    smoothing_iter(7),
    diffuse_smooth_weights_iter(6),
    smooth_force_a(0.5f),
    smooth_force_b(0.5f),
    smooth_smear(0.f),
    d_input_smooth_factors(_mesh->get_nb_vertices()),
    d_smooth_factors_conservative(_mesh->get_nb_vertices(), 0.f),
    d_smooth_factors_laplacian(_mesh->get_nb_vertices()),
    d_input_vertices(_mesh->get_nb_vertices()),
    d_edge_lengths(_mesh->get_nb_edges()),
    d_edge_mvc(_mesh->get_nb_edges()),
    d_vertices_state(_mesh->get_nb_vertices()),
    d_vertices_states_color(EAnimesh::NB_CASES),
    h_junction_radius(_skel->nb_joints()),
//    d_input_normals(m->get_nb_vertices()),
    d_output_vertices(_mesh->get_nb_vertices()),
    d_output_normals(_mesh->get_nb_vertices()),
    d_ssd_normals(_mesh->get_nb_vertices()),
    d_ssd_vertices(_mesh->get_nb_vertices()),
    d_gradient(_mesh->get_nb_vertices()),
    d_input_tri(_mesh->get_nb_tri()*3),
    d_edge_list(_mesh->get_nb_edges()),
    d_edge_list_offsets(2 * _mesh->get_nb_vertices()),
    d_joints(), d_weights(),
    d_jpv(2 * _mesh->get_nb_vertices()),
    h_weights(_mesh->get_nb_vertices()),
    d_base_potential(_mesh->get_nb_vertices()),
    d_base_gradient(_mesh->get_nb_vertices()),
    d_piv(_mesh->get_nb_faces()),
    d_packed_vert_map(_mesh->get_nb_vertices()),
    d_unpacked_normals(_mesh->get_nb_vertices() * _mesh->_max_faces_per_vertex),
    d_unpacked_tangents(_mesh->get_nb_vertices() * _mesh->_max_faces_per_vertex),
    d_rot_axis(_mesh->get_nb_vertices()),
    d_ssd_interpolation_factor(_mesh->get_nb_vertices(), 0.f),
    h_vertices_nearest_bones(_mesh->get_nb_vertices()),
    d_vertices_nearest_bones(_mesh->get_nb_vertices()),
    nb_vertices_by_bones(_skel->get_bones().size()),
    h_vertices_nearest_joint(_mesh->get_nb_vertices()),
    d_vertices_nearest_joint(_mesh->get_nb_vertices()),
    d_nearest_bone_in_device_mem(_mesh->get_nb_vertices()),
    d_nearest_joint_in_device_mem(_mesh->get_nb_vertices()),
    h_nearest_bone_dist(_mesh->get_nb_vertices()),
    vmap_old_new(_mesh->get_nb_vertices()),
    vmap_new_old(_mesh->get_nb_vertices()),
    d_rear_verts(_mesh->get_nb_vertices()),
    h_half_angles(_skel->nb_joints()),
    d_half_angles(_skel->nb_joints()),
    h_orthos(_skel->nb_joints()),
    d_orthos(_skel->nb_joints()),
    d_flip_propagation(_mesh->get_nb_vertices()),
    h_vert_buffer(_mesh->get_nb_vertices()),
    d_vert_buffer(_mesh->get_nb_vertices()),
    d_vert_buffer_2(_mesh->get_nb_vertices()),
    d_vals_buffer(_mesh->get_nb_vertices())
{

    // Compute nearest bone and nearest joint from each vertices
    clusterize();

    int nb_vert = _mesh->get_nb_vertices();
    Host::Array<EAnimesh::Vert_state> h_vert_state(nb_vert);
    for (int i = 0; i < nb_vert; ++i)
    {
        vmap_old_new[i] = i;
        vmap_new_old[i] = i;
        h_vert_state[i] = EAnimesh::NOT_DISPLACED;
    }

    d_vertices_state.copy_from(h_vert_state);

    d_vertices_states_color.set(EAnimesh::POTENTIAL_PIT      , make_float4(1.f, 0.f, 1.f, 0.99f)); // purple
    d_vertices_states_color.set(EAnimesh::GRADIENT_DIVERGENCE, make_float4(1.f, 0.f, 0.f, 0.99f)); // red
    d_vertices_states_color.set(EAnimesh::NB_ITER_MAX        , make_float4(0.f, 0.f, 1.f, 0.99f)); // blue
    d_vertices_states_color.set(EAnimesh::NOT_DISPLACED      , make_float4(1.f, 1.f, 0.f, 0.99f)); // yellow
    d_vertices_states_color.set(EAnimesh::FITTED             , make_float4(0.f, 1.f, 0.f, 0.99f)); // green
    d_vertices_states_color.set(EAnimesh::OUT_VERT           , make_float4(1.f, 1.f, 1.f, 0.99f)); // white
    d_vertices_states_color.set(EAnimesh::NORM_GRAD_NULL     , make_float4(0.f, 0.f, 0.f, 0.99f)); // black

    // Not mandatory but it is supposed to accelerate a little bit animation
    // when activated

    // Fill the attributes in device memory
    copy_mesh_data(*_mesh);

    set_default_bones_radius();

    init_smooth_factors(d_input_smooth_factors);
    init_rigid_ssd_weights();
    init_ssd_interpolation_weights();

    compute_mvc();

    update_base_potential();
}

// -----------------------------------------------------------------------------

void Animesh::init_verts_per_bone()
{
    std::vector< std::vector<Vec3_cu> >& vertices = h_input_verts_per_bone;
    std::vector< std::vector<Vec3_cu> >& normals  = h_input_normals_per_bone;
    std::vector< std::vector<int>     >& vert_ids = h_verts_id_per_bone;
    vertices.clear();
    normals. clear();
    vertices.resize(_skel->nb_joints());
    vert_ids.resize(_skel->nb_joints());
    normals. resize(_skel->nb_joints());

    for(int i = 0; i < _mesh->get_nb_vertices(); i++)
    {
        int nearest = h_vertices_nearest_bones[i];

        if(_mesh->is_disconnect(i))
            continue;

        const Vec3_cu vert = _mesh->get_vertex(i);
        const Vec3_cu norm = _mesh->get_normal(i);

        vertices[nearest].push_back( Vec3_cu(vert.x,  vert.y,  vert.z)              );
        normals [nearest].push_back( Vec3_cu(norm.x,  norm.y,  norm.z).normalized() );

        vert_ids[nearest].push_back( i );
    }
}

// -----------------------------------------------------------------------------

Animesh::~Animesh()
{
}

// -----------------------------------------------------------------------------

void Animesh::init_vert_to_fit()
{
    assert(d_ssd_interpolation_factor.size() > 0);

    Cuda_utils::HA_float ssd_factor(d_ssd_interpolation_factor.size());
    ssd_factor.copy_from(d_ssd_interpolation_factor);

    int nb_vert = _mesh->get_nb_vertices();
    std::vector<int> h_vert_to_fit_base;
    h_vert_to_fit_base.reserve(nb_vert);
    int acc = 0;
    for (int i = 0; i < nb_vert; ++i)
    {
        if( !_mesh->is_disconnect(i) && ssd_factor[i] < (1.f - 0.00001f) ){
            h_vert_to_fit_base.push_back( i );
            acc++;
        }
    }

    d_vert_to_fit.     malloc(acc);
    d_vert_to_fit_base.malloc(acc);

    d_vert_to_fit_buff_scan.malloc(acc+1);
    d_vert_to_fit_buff.malloc(acc);
    h_vert_to_fit_buff.malloc(acc);
    h_vert_to_fit_buff_2.malloc(acc);

    d_vert_to_fit_base.copy_from(h_vert_to_fit_base);
    d_vert_to_fit.     copy_from(h_vert_to_fit_base);
}

// -----------------------------------------------------------------------------

void Animesh::copy_mesh_data(const Mesh& a_mesh)
{
    const int nb_vert = a_mesh.get_nb_vertices();

    const Mesh::Packed_data* d = a_mesh.get_packed_vert_map();
    Cuda_utils::mem_cpy_htd(d_packed_vert_map.ptr(), d, nb_vert);

    Host::Array<Point_cu > input_vertices(nb_vert);
    Host::Array<Vec3_cu>   input_normals (nb_vert);
    Host::Array<bool>      flip_prop     (nb_vert);
    for(int i = 0; i < nb_vert; i++)
    {
        Point_cu  pos = Convs::to_point( a_mesh.get_vertex(i) );

        input_vertices[i] = pos;
        input_normals [i] = a_mesh.get_normal(i);
        flip_prop     [i] = false;
    }

    int n_faces = a_mesh.get_nb_faces();
    Host::Array<Mesh::PrimIdxVertices> h_piv(n_faces);
    for(int i = 0; i < n_faces; i++){
        h_piv[i] = a_mesh.get_piv(i);
    }
    d_piv.copy_from(h_piv);

    d_input_vertices.copy_from(input_vertices);
//    d_input_normals.copy_from(input_normals);
    d_ssd_normals.copy_from(input_normals);
    d_flip_propagation.copy_from(flip_prop);

    HA_int h_edge_list(a_mesh.get_nb_edges());
    HA_int h_edge_list_offsets(2*nb_vert);
    for(int i = 0; i < a_mesh.get_nb_edges(); i++){
        h_edge_list[i] = a_mesh.get_edge(i);
    }
    for(int i = 0; i < nb_vert; i++){
        h_edge_list_offsets[2*i  ] = a_mesh.get_edge_offset(2*i  );
        h_edge_list_offsets[2*i+1] = a_mesh.get_edge_offset(2*i+1);
    }
    d_edge_list.copy_from(h_edge_list);
    d_edge_list_offsets.copy_from(h_edge_list_offsets);

    Cuda_utils::mem_cpy_htd(d_input_tri. ptr(), a_mesh._tri , a_mesh._nb_tri*3 );
}

// -----------------------------------------------------------------------------



void Animesh::compute_mvc()
{
    //Device::Array<Vec3_cu> d_grad( d_input_vertices.size() );
    Host::Array<float> edge_lengths(_mesh->get_nb_edges());
    Host::Array<float> edge_mvc    (_mesh->get_nb_edges());
    for(int i = 0; i < _mesh->get_nb_vertices(); i++)
    {
        Point_cu pos = Convs::to_point( _mesh->get_vertex(i)      );
        Vec3_cu  nor = Convs::to_point( _mesh->get_mean_normal(i) ); // FIXME : should be the gradient

        Mat3_cu frame = Mat3_cu::coordinate_system( nor ).transpose();
        float sum = 0.f;
        bool  out = false;
        // Look up neighborhood
        int dep      = _mesh->get_edge_offset(i*2    );
        int nb_neigh = _mesh->get_edge_offset(i*2 + 1);
        int end      = (dep+nb_neigh);

        if( nor.norm() < 0.00001f || _mesh->is_vert_on_side(i) ) {
            for(int n = dep; n < end; n++) edge_mvc[n] = 0.f;
        }
        else
        {
            for(int n = dep; n < end; n++)
            {
                int id_curr = _mesh->get_edge( n );
                int id_next = _mesh->get_edge( (n+1) >= end  ? dep   : n+1 );
                int id_prev = _mesh->get_edge( (n-1) <  dep  ? end-1 : n-1 );

                // compute edge length
                Point_cu  curr = Convs::to_point( _mesh->get_vertex(id_curr) );
                Vec3_cu e_curr = (curr - pos);
                edge_lengths[n] = e_curr.norm();

                // compute mean value coordinates
                // coordinates are computed by projecting the neighborhood to the
                // tangent plane
                {
                    // Project on tangent plane
                    Vec3_cu e_next = Convs::to_point( _mesh->get_vertex(id_next) ) - pos;
                    Vec3_cu e_prev = Convs::to_point( _mesh->get_vertex(id_prev) ) - pos;

                    e_curr = frame * e_curr;
                    e_next = frame * e_next;
                    e_prev = frame * e_prev;

                    e_curr.x = 0.f;
                    e_next.x = 0.f;
                    e_prev.x = 0.f;

                    float norm_curr_2D = e_curr.norm();

                    e_curr.normalize();
                    e_next.normalize();
                    e_prev.normalize();

                    // Computing mvc
                    float anext = std::atan2( -e_prev.z * e_curr.y + e_prev.y * e_curr.z, e_prev.dot(e_curr) );
                    float aprev = std::atan2( -e_curr.z * e_next.y + e_curr.y * e_next.z, e_curr.dot(e_next) );

                    float mvc = 0.f;
                    if(norm_curr_2D > 0.0001f)
                        mvc = (std::tan(anext*0.5f) + std::tan(aprev*0.5f)) / norm_curr_2D;

                    sum += mvc;
                    edge_mvc[n] = mvc;
                    out = out || mvc < 0.f;
                }
            }
            // we ignore points outside the convex hull
            if( sum  <= 0.f || out || isnan(sum) ) {
                for(int n = dep; n < end; n++) edge_mvc[n] = 0.f;
            }
        }

    }
    d_edge_lengths.copy_from( edge_lengths );
    d_edge_mvc.    copy_from( edge_mvc     );
}

void Animesh::init_ssd_interpolation_weights()
{
    int n = d_input_vertices.size();

//    Host::Array<float> base_potential(n);
//    base_potential.copy_from(d_base_potential);

    Host::Array<float> base_ssd_weights(n);
    base_ssd_weights.copy_from(d_ssd_interpolation_factor);
    //_mesh->diffuse_along_mesh(base_ssd_weights.ptr(), 1.f, 2);

    d_ssd_interpolation_factor.copy_from(base_ssd_weights);

    init_vert_to_fit();
}

// -----------------------------------------------------------------------------

float Animesh::compute_nearest_vert_to_bone(int bone_id)
{
    return 1.f;
}

// -----------------------------------------------------------------------------

void Animesh::clusterize_euclidean(HA_int& vertices_nearest_bones,
                                   HA_int& h_vertices_nearest_joint,
                                   HA_int& nb_vert_by_bone)
{
    const int nb_bones = _skel->get_bones().size();
    assert(nb_vert_by_bone.size() == nb_bones);
    for(int i = 0; i<nb_bones; i++)
        nb_vert_by_bone[i] = 0;

    int n = _mesh->get_nb_vertices();
    for(int i = 0; i < n ; i++)
    {
        float d0  = std::numeric_limits<float>::infinity();
        int   nd0 = _skel->root();

        float joint_dist       = std::numeric_limits<float>::infinity();
        int   nearest_joint_id = _skel->root();

        const Point_cu current_vertex = _mesh->get_vertex(i).to_point();
        for(int j = 0; j < _skel->nb_joints(); j++)
        {
            const Bone* b = _skel->get_bone( j );

            if( _skel->is_leaf(j) )
                continue;

            // Compute nearest bone
            float dist2 = b->dist_sq_to(current_vertex);

            if(dist2 <= d0){
                d0  = dist2;
                nd0 = j;
            }

            // compute nearest joint
            const Point_cu joint = _skel->joint_pos(j).to_point();
            const Vec3_cu dir    = current_vertex-joint;
            float dist = dir.norm();
            // works fine but some mesh have corrupted normals so for the moment
            // I don't use this information
            float sign = 1.f;// dir.dot( current_normal );
            if(dist < joint_dist && sign >= 0){
                nearest_joint_id = j;
                joint_dist       = dist;
            }
        }
        h_nearest_bone_dist     [i] = sqrt(d0);
        vertices_nearest_bones  [i] = nd0;
        h_vertices_nearest_joint[i] = nearest_joint_id;
        nb_vert_by_bone[nd0]++;
    }
}

// -----------------------------------------------------------------------------

void Animesh::clusterize(int n_voxels)
{
    clusterize_euclidean(h_vertices_nearest_bones, h_vertices_nearest_joint, nb_vertices_by_bones);

    init_verts_per_bone();
    update_nearest_bone_joint_in_device_mem();
}

// -----------------------------------------------------------------------------

void Animesh::update_nearest_bone_joint_in_device_mem()
{
    int n = _mesh->get_nb_vertices();
    // Convert host ids to device ids for the nearest joints
    std::vector<DBone_id> tmp (n);
    std::vector<DBone_id> tmp2(n);
    for(int i = 0; i < n; i++){
        tmp [i] = _skel->get_bone_didx( h_vertices_nearest_bones[i] );
        tmp2[i] = _skel->get_bone_didx( h_vertices_nearest_joint[i] );
    }
    d_nearest_bone_in_device_mem. copy_from(tmp);
    d_nearest_joint_in_device_mem.copy_from(tmp2);

    d_vertices_nearest_bones.copy_from(h_vertices_nearest_bones);
    d_vertices_nearest_joint.copy_from(h_vertices_nearest_joint);
}

// -----------------------------------------------------------------------------

void Animesh::set_default_bones_radius()
{
    const int nb_verts  = _mesh->get_nb_vertices();
    const int nb_joints = _skel->nb_joints();

    std::vector<float> avg_rad     (nb_joints);
    std::vector<float> nearest_rad (nb_joints);
    std::vector<float> farthest_rad(nb_joints);
    std::vector<int>   nb_smp      (nb_joints);

    const float inf = std::numeric_limits<float>::infinity();
    for(int i = 0; i < nb_joints; i++) {
        nearest_rad [i] = inf;
        farthest_rad[i] = 0.f;
        avg_rad     [i] = 0.f;
        nb_smp      [i] = 0;
    }

    for(int i = 0; i < nb_verts; i++)
    {
        const int j = h_vertices_nearest_bones[i];
        const Point_cu vert = _mesh -> get_vertex(i).to_point();
        float  d = _skel->get_bone(j)->dist_to( vert );

        nearest_rad [j] = std::min(nearest_rad [j], d);
        farthest_rad[j] = std::max(farthest_rad[j], d);
        avg_rad[j] += d;
        nb_smp[j]++;
    }

    for(int i = 0; i < nb_joints; i++)
    {
        // Cylinder radius is average vertices distance
        avg_rad[i] = nb_smp[i] ? avg_rad[i] / nb_smp[i] : 1.f;
        _skel->set_bone_radius(i, avg_rad[i]);

        // HRBF compact support radius is farthest vertex distance
        const float radius = farthest_rad[i] == 0.f ? 1.f : farthest_rad[i];
        _skel->set_bone_hrbf_radius(i, radius);

        // Junction radius is nearest vertex distance
        h_junction_radius[i] = nearest_rad[i] == inf ? 1.f : nearest_rad[i];
    }
}

// -----------------------------------------------------------------------------

float distsqToSeg(const Point_cu& v, const Point_cu& p1, const Point_cu& p2)
{
    Vec3_cu dir   = p2 - p1;
    Vec3_cu difp2 = p2 - v;

    if(difp2.dot(dir) < 0.f) return difp2.norm_squared();

    Vec3_cu difp1 = v - p1;
    float dot = difp1.dot(dir);

    if(dot <= 0.f) return difp1.norm_squared();

    return fmax(0.f, difp1.norm_squared() - dot*dot / dir.norm_squared());
}

// -----------------------------------------------------------------------------


Point_cu projToSeg(const Point_cu& v, const Point_cu& p1, const Point_cu& p2)
{

  Vec3_cu dir = p2 - p1;

  if( (p2 - v).dot(dir) < 0.f) return p2;

  float dot = (v - p1).dot(dir);

  if(dot <= 0.f) return p1;

  return p1 + dir * (dot / dir.norm_squared());
}

// -----------------------------------------------------------------------------

bool vectorInCone(const Vec3_cu& v, const std::vector<Vec3_cu>& ns)
{
    int i;
    Vec3_cu avg = Vec3_cu(0.f, 0.f, 0.f);
    for(i = 0; i < (int)ns.size(); ++i)
        avg += ns[i];

    return v.normalized().dot(avg.normalized()) > 0.5f;
}

void Animesh::init_rigid_ssd_weights()
{
    int nb_vert = _mesh->get_nb_vertices();

    std::vector<float> weights(nb_vert);
    std::vector<int>   joints (nb_vert);
    Host::Array<int>   jpv    (2u*nb_vert);

    for(int i = 0; i < nb_vert; ++i)
    {
        joints [i] = h_vertices_nearest_bones[i];
        weights[i] = 1.f;

        jpv[i*2    ] = i; // starting index
        jpv[i*2 + 1] = 1; // number of bones influencing the vertex

        int start = i;
        int end   = start + 1;

        h_weights[i].clear();
        for(int j = start; j < end ; j++)
            h_weights[i][joints[j]] = weights[j];
    }

    d_jpv.copy_from(jpv);
    d_weights.malloc(nb_vert);
    d_weights.copy_from(weights);
    d_joints.malloc(nb_vert);
    d_joints.copy_from(joints);
}


void Animesh::init_smooth_factors(Cuda_utils::DA_float& d_smooth_factors)
{
    const int nb_vert = _mesh->get_nb_vertices();
    HA_float smooth_factors(nb_vert);

    for(int i=0; i<nb_vert; i++)
        smooth_factors[i] = 0.0f;

    d_smooth_factors.copy_from(smooth_factors);
}

// -----------------------------------------------------------------------------

void Animesh::diffuse_attr(int nb_iter, float strength, float *attr)
{
    Animesh_kers::diffuse_values(attr,
                            d_vals_buffer.ptr(),
                            d_edge_list,
                            d_edge_list_offsets,
                            strength,
                            nb_iter);
}

// -----------------------------------------------------------------------------

void Animesh::get_anim_vertices_aifo(std::vector<float>& anim_vert)
{
    const int nb_vert = d_output_vertices.size();
    anim_vert.reserve(nb_vert);
    Cuda_utils::HA_Point_cu h_out_verts(nb_vert);
    h_out_verts.copy_from(d_output_vertices);

    for(int i = 0; i < nb_vert; i++)
    {
        Point_cu p = h_out_verts[vmap_new_old[i]];
        anim_vert.push_back(p.x);
        anim_vert.push_back(p.y);
        anim_vert.push_back(p.z);
    }
}

// -----------------------------------------------------------------------------

void Animesh::set_bone_type(int id, int bone_type)
{
    _skel->reset();
    Bone* bone = 0;
    const Bone* prev_bone = _skel->get_bone( id );
    float rad = prev_bone->radius();
    switch(bone_type){
    case EBone::PRECOMPUTED:
    {
        // We don't precompute an already precomputed primitive
        assert(_skel->bone_type(id) != EBone::PRECOMPUTED );
        // Precompute a SSD bone is useless and should be forbiden
        assert(_skel->bone_type(id) != EBone::SSD         );

        Bone_precomputed* b = new Bone_precomputed( prev_bone->get_obbox() );
        b->get_primitive().fill_grid_with( _skel->get_bone(id) );
        bone = b;
        break;
    }
    case EBone::HRBF:     bone = new Bone_hrbf(rad);      break;
    case EBone::SSD:      bone = new Bone_ssd();          break;

    default: //unknown bone type !
        assert(false);
        break;

    }


    bone->set_radius(rad);
    _skel->set_bone(id, bone);

    init_ssd_interpolation_weights();
    _skel->unreset();
}

// -----------------------------------------------------------------------------

float Animesh::get_junction_radius(int bone_id){
    assert(bone_id >=0                 );
    assert(bone_id < _skel->nb_joints());
    return h_junction_radius[bone_id];
}

// -----------------------------------------------------------------------------

void Animesh::set_junction_radius(int bone_id, float rad)
{
    assert(bone_id >=0                 );
    assert(bone_id < _skel->nb_joints());
    h_junction_radius[bone_id] = rad;
}

// -----------------------------------------------------------------------------

void Animesh::set_ssd_weight(int id_vertex, int id_joint, float weight)
{
    id_joint = _skel->parent(id_joint);

    assert(id_vertex < (int)d_input_vertices.size());
    // clamp [0, 1]
    weight = fmax(0.f, fmin(weight, 1.f));

    float old_weight = get_ssd_weight(id_vertex, id_joint);
    float delta      = old_weight - weight;

    int start, end;
    d_jpv.fetch(id_vertex*2  , start);
    d_jpv.fetch(id_vertex*2+1, end  );

    delta = delta / (float)(end-1);

    for(int i=start; i<(start+end); i++)
    {
        int current_joint;
        d_joints.fetch(i, current_joint);
        if(current_joint == id_joint)
            d_weights.set(i, weight);
        else
        {
            float w;
            d_weights.fetch(i, w);
            d_weights.set(i, w+delta);
        }
    }
}

// -----------------------------------------------------------------------------

float Animesh::get_ssd_weight(int id_vertex, int id_joint)
{
    assert(id_vertex < d_input_vertices.size());

    int start, end;
    d_jpv.fetch(id_vertex*2  , start);
    d_jpv.fetch(id_vertex*2+1, end  );

    for(int i=start; i<(start+end); i++)
    {
        int current_joint;
        d_joints.fetch(i, current_joint);
        if(current_joint == id_joint)
        {
            float w;
            d_weights.fetch(i, w);
            return w;
        }
    }

    // Joint "id_joint" is not associated to this vertex
    assert(false);
    return 0.f;
}

// -----------------------------------------------------------------------------

void Animesh::get_ssd_weights(std::vector<std::map<int, float> >& weights)
{
    const int nb_vert = d_input_vertices.size();
    weights.clear();
    weights.resize(nb_vert);

    HA_float h_weights(d_weights.size());
    HA_int h_joints(d_joints.size());
    HA_int h_jpv(d_jpv.size());
    h_weights.copy_from(d_weights);
    h_joints.copy_from(d_joints);
    h_jpv.copy_from(d_jpv);

    for( int i = 0; i < nb_vert; i++)
    {
        int start = h_jpv[i*2];
        int end   = start + h_jpv[i*2 + 1];
        weights[i].clear();
        for(int j = start; j < end ; j++){
            weights[i][h_joints[j]] = h_weights[j];
            //std::cout << h_weights[j] << std::endl;
        }
    }
}

// -----------------------------------------------------------------------------

void Animesh::update_host_ssd_weights()
{
    get_ssd_weights(h_weights);
}

// -----------------------------------------------------------------------------

void Animesh::set_ssd_weights(const std::vector<std::map<int, float> >& in_weights)
{
    const int nb_vert = d_input_vertices.size();
    assert( in_weights.size() == (unsigned)nb_vert );

    std::vector<float> weights;
    std::vector<int>   joints;
    HA_int             jpv(nb_vert*2);

    weights.reserve(nb_vert*2);
    joints.reserve(nb_vert*2);

    int acc = 0;
    for( int i = 0; i < nb_vert; i++)
    {
        const std::map<int, float>& map = in_weights[i];
        jpv[i*2    ] = acc;
        jpv[i*2 + 1] = map.size();
        std::map<int, float>::const_iterator it;
        for(it = map.begin(); it != map.end(); ++it)
        {
            joints.push_back(it->first);
            weights.push_back(it->second);
        }
        acc += map.size();
    }

    d_weights.malloc(weights.size());
    d_joints.malloc(joints.size());
    d_jpv.malloc(jpv.size());
    d_weights.copy_from(weights);
    d_joints.copy_from(joints);
    d_jpv.copy_from(jpv);
}

// -----------------------------------------------------------------------------

void Animesh::update_device_ssd_weights()
{
    set_ssd_weights(h_weights);
}

// -----------------------------------------------------------------------------

void Animesh::export_weights(const char* filename)
{
    using namespace std;
    ofstream file(filename, ios_base::out|ios_base::trunc);

    if(!file.is_open()){
        cerr << "Error exporting file " << filename << endl;
        exit(1);
    }

    // Copy to host :
    HA_int   h_jpv(d_jpv.size());
    HA_int   h_joints(d_joints.size());
    HA_float h_weights(d_weights.size());

    h_jpv.copy_from(d_jpv);
    h_joints.copy_from(d_joints);
    h_weights.copy_from(d_weights);

    for(int i = 0; i < d_input_vertices.size(); i++)
    {
        int start, end;
        float sum_weights = 0.f;
        // vertices are not necessarily
        start = h_jpv[vmap_new_old[i]*2    ];
        end   = h_jpv[vmap_new_old[i]*2 + 1];

        for(int j=start; j<(start+end); j++)
        {
            float weight = h_weights[j];
            int   bone   = h_joints[j];
            sum_weights += weight;

            file << bone << " " << weight << " ";
        }

        if((sum_weights > 1.0001f) || (sum_weights < -0.0001f)){
            std::cerr << "WARNING: exported ssd weights does not sum to one ";
            std::cerr << "(line " << (i+1) << ")" << std::endl;
        }
        file << endl;
    }
    file.close();
}


// -----------------------------------------------------------------------------

void Animesh::read_weights_from_file(const char* filename,
                                          bool file_has_commas)
{
    using namespace std;
    using namespace Cuda_utils;

    ifstream file(filename);

    int n = _mesh -> get_nb_vertices();
    std::vector<float> h_weights; h_weights.reserve(2*n);
    std::vector<int>   h_joints; h_joints.reserve(2*n);
    Host::Array<int>   h_jpv(2*n);

    if(!file.is_open()){
        cerr << "Error opening file: " << filename << endl;
        exit(1);
    }

    int k = 0;
    for(int i = 0; i < n; i++)
    {
        std::string str_line;
        std::getline(file, str_line);
        std::stringbuf current_line_sb(str_line, ios_base::in);

        istream current_line(&current_line_sb);
        int j = 0;
        //int j_old = -1;
        float weight, sum_weights = 0.f;
        int p = 0;
        while(!current_line.eof() && !str_line.empty())
        {
            current_line >> j;
            //if(j == j_old) break;
            if(file_has_commas) current_line.ignore(1,',');

            current_line >> weight;

            if(file_has_commas) current_line.ignore(1,',');

            current_line.ignore(10,' ');

            if(current_line.peek() == '\r') current_line.ignore(1,'\r');

            if(current_line.peek() == '\n') current_line.ignore(1,'\n');

            p++;
            h_weights.push_back( weight ); // SSD weight
            h_joints. push_back(  j     ); // joint number
            //j_old = j;

            sum_weights += weight;
            if(j < 0 || j > _skel->nb_joints()){
                std::cerr << "ERROR: incorrect joint id in imported ssd weights.";
                std::cerr << "Maybe the file does not match the skeleton";
                std::cerr << std::endl;
            }
        }

        if((sum_weights > 1.0001f) || (sum_weights < -0.0001f)){
            std::cerr << "WARNING: imported ssd weights does not sum to one ";
            std::cerr << "(line " << (i+1) << ")" << std::endl;
        }

        // we use vmap_old_new because Animesh does not necessarily
        // stores vertices in the same order as in the off file
        h_jpv[2*vmap_old_new[i]  ] = k; //read start position
        h_jpv[2*vmap_old_new[i]+1] = p; //number of joints modifying that vertex
        k += p;
    } // END FOR NB lINES

    // Copy weights to device mem
    d_jpv.copy_from(h_jpv);
    d_weights.malloc(k);
    d_weights.copy_from(h_weights);
    d_joints.malloc(k);
    d_joints.copy_from(h_joints);

    update_host_ssd_weights();

    cout << "file \"" << filename << "\" loaded successfully" << endl;
    set_default_bones_radius();
    file.close();
}

// -----------------------------------------------------------------------------

int Animesh::pack_vert_to_fit(Cuda_utils::Host::Array<int>& in,
                                   Cuda_utils::Host::Array<int>& out,
                                   int size)
{
    Cuda_utils::mem_cpy_dth(in.ptr(), d_vert_to_fit.ptr(), size);

    int j = 0;
    for(int i = 0; i < size; i++)
    {
        int elt = in[i];
        if(elt != -1)
        {
            out[j] = elt;
            j++;
        }
    }

    Cuda_utils::mem_cpy_htd(d_vert_to_fit.ptr(), out.ptr(), j);
    return j;
}

// -----------------------------------------------------------------------------

#include "cuda_utils_thrust.hpp"

__global__ static
void transform_vert_to_fit(const int* src, int* dst, const int nb_vert)
{
    const int p = blockIdx.x * blockDim.x + threadIdx.x;
    if(p < nb_vert) dst[p] = src[p] < 0 ? 0 : 1;
}

/// here src must be different from dst
__global__ static
void pack(const int* prefix_sum, const int* src, int* dst, const int nb_vert)
{
    const int p = blockIdx.x * blockDim.x + threadIdx.x;
    if(p < nb_vert){
        const int elt = src[p];
        if(elt >= 0) dst[ prefix_sum[p] ] = elt;
    }
}

int Animesh::pack_vert_to_fit_gpu(
        Cuda_utils::Device::Array<int>& d_vert_to_fit,
        Cuda_utils::Device::Array<int>& buff,
        Cuda_utils::Device::Array<int>& packed_array,
        int nb_vert_to_fit)
{
    if(nb_vert_to_fit == 0) return 0;
    assert(d_vert_to_fit.size() >= nb_vert_to_fit           );
    assert(buff.size()          >= d_vert_to_fit.size() + 1 );
    assert(packed_array.size()  >= d_vert_to_fit.size()     );

    const int block_s = 16;
    const int grid_s  = (nb_vert_to_fit + block_s - 1) / block_s;
    transform_vert_to_fit<<<grid_s, block_s >>>(d_vert_to_fit.ptr(), buff.ptr()+1, nb_vert_to_fit);
    buff.set(0, 0);// First element to zero
    CUDA_CHECK_ERRORS();

    // Compute prefix sum in buff between [1 nb_vert_to_fit]
    Cuda_utils::inclusive_scan(0, nb_vert_to_fit-1, buff.ptr()+1);

    const int new_nb_vert_to_fit = buff.fetch(nb_vert_to_fit);

    pack<<<grid_s, block_s >>>(buff.ptr(), d_vert_to_fit.ptr(), packed_array.ptr(), nb_vert_to_fit);
    CUDA_CHECK_ERRORS();

    //Cuda_utils::mem_cpy_dtd(d_vert_to_fit.ptr(), packed_array.ptr(), new_nb_vert_to_fit);

    return new_nb_vert_to_fit;
}

void Animesh::reset_flip_propagation(){
    int nb_vert = _mesh->get_nb_vertices();
    Host::Array<bool> flip_prop(nb_vert);
    for(int i = 0; i < nb_vert; i++){
        flip_prop[i] = false;
    }

    d_flip_propagation.copy_from(flip_prop);
}
