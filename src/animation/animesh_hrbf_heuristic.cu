#include "hip/hip_runtime.h"
/*
 Implicit skinning
 Copyright (C) 2013 Rodolphe Vaillant, Loic Barthe, Florian Cannezin,
 Gael Guennebaud, Marie Paule Cani, Damien Rohmer, Brian Wyvill,
 Olivier Gourmel

 This program is free software: you can redistribute it and/or modify
 it under the terms of the GNU General Public License 3 as published by
 the Free Software Foundation.

 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.

 You should have received a copy of the GNU General Public License
 along with this program. If not, see <http://www.gnu.org/licenses/>
 */
#include "animesh_hrbf_heuristic.hpp"
#include "animesh.hpp"

// -----------------------------------------------------------------------------

#include "timer.hpp"
#include "conversions.hpp"
#include "utils_sampling.hpp"
#include "skeleton.hpp"
#include "hrbf_env.hpp"

// -----------------------------------------------------------------------------
namespace { __device__ void fix_debug() { } }

#include <iostream>
#include <limits>
#include <hip/hip_runtime.h>

using namespace Cuda_utils;
using namespace HRBF_env;

HRBF_sampling::HRBF_sampling(const Mesh *mesh_, const Skeleton *skel_, const VertToBoneInfo &vertToBoneInfo_):
    _bone_id(-1),
    _factor_siblings(false),
    vertToBoneInfo(vertToBoneInfo_),
    mesh(mesh_),
    skel(skel_)
{ }

void HRBF_sampling::factor_samples(std::vector<int>& vert_ids,
                               std::vector<Vec3_cu>& vertices,
                               std::vector<Vec3_cu>& normals) const
{
    vert_ids.clear();
    vertices.clear();
    normals. clear();

    int parent = skel->parent( _bone_id );
    std::vector<int> dummy(1, _bone_id);
    const std::vector<int>& sons = parent == -1 ? dummy : skel->get_sons(parent);
    assert(sons.size() > 0);

    if(_factor_siblings && sons[0] == _bone_id)
        return;

    for( unsigned i = 0; i < sons.size(); i++)
    {
        const int bone_id = _factor_siblings ? sons[i] : _bone_id;
        const auto &ids   = vertToBoneInfo.h_verts_id_per_bone.at(bone_id);
        const auto &nors  = vertToBoneInfo.h_input_normals_per_bone.at(bone_id);
        const auto &verts = vertToBoneInfo.h_input_verts_per_bone.at(bone_id);

        vert_ids.insert(vert_ids.end(), ids.begin(), ids.end());
        vertices.insert(vertices.end(), verts.begin(), verts.end());
        normals.insert(normals.end(), nors.begin(), nors.end());

        if( !_factor_siblings ) break;
    }
}

// -----------------------------------------------------------------------------

void HRBF_sampling::clamp_samples(std::vector<int>& vert_ids_,
                              std::vector<Vec3_cu>& verts_,
                              std::vector<Vec3_cu>& normals_) const
{
    std::vector<int> vert_ids;
    std::vector<Vec3_cu> verts;
    std::vector<Vec3_cu> normals;

    vert_ids.reserve( vert_ids_.size() );
    verts.   reserve( verts_.size()    );
    normals. reserve( normals_.size()  );

    for(unsigned id = 0; id < verts_.size(); id++)
    {
        const int nearest_bone = vertToBoneInfo.h_vertices_nearest_bones[ vert_ids_[id] ];
        const Bone* b = skel->get_bone(nearest_bone);
        const float length = b->length();

        const Point_cu vert = Convs::to_point(verts_[id]);
        const float dist_proj = b->dist_proj_to(vert);

        Vec3_cu dir_proj = vert - (b->org() + b->dir().normalized() * dist_proj);

        float jlength = length * _jmax;
        float plength = length * _pmax;

        const Vec3_cu normal = normals_[id];

        const std::vector<int>& sons = skel->get_sons(nearest_bone);
        bool leaf = sons.size() > 0 ? skel->is_leaf(sons[0]) : true;

        if( (dist_proj >= -plength ) &&
            (dist_proj < (length + jlength) || leaf) &&
            dir_proj.dot(normal) >= _fold )
        {
            verts.   push_back( verts_   [id] );
            vert_ids.push_back( vert_ids_[id] );
            normals. push_back( normal        );
        }
    }

    vert_ids_.swap( vert_ids );
    verts_.   swap( verts    );
    normals_. swap( normals  );
}

// -----------------------------------------------------------------------------

void Adhoc_sampling::sample(std::vector<Vec3_cu>& out_verts,
                        std::vector<Vec3_cu>& out_normals) const
{
    std::vector<Vec3_cu> in_verts;
    std::vector<int>     in_vert_ids;
    std::vector<Vec3_cu> in_normals;
    factor_samples(in_vert_ids, in_verts, in_normals);

    std::vector<bool> done;
    done.resize(in_verts.size(), false);
    for(unsigned id = 0; id < in_verts.size(); id++)
    {

        const Bone* b = skel->get_bone(_bone_id);
        float length = b->length();

        Point_cu vert = Convs::to_point(in_verts[id]);
        float dist_proj = b->dist_proj_to(vert);

        Vec3_cu dir_proj = vert - (b->org() + b->dir().normalized() * dist_proj);

        float jlength = length * _jmax;
        float plength = length * _pmax;

        Vec3_cu normal = in_normals[id];
        if(dist_proj >= -jlength && dist_proj < (length + plength) &&
                dir_proj.dot(normal) >= _fold )
        {
            // Check for to close samples
            float dist = std::numeric_limits<float>::infinity();
            for(unsigned j = 0; j < in_verts.size(); j++)
            {
                float norm = (Convs::to_vector(vert) - in_verts[j]).norm();
                if( (unsigned)id != j && !done[j] && norm < dist)
                    dist = norm;
            }

            if(dist > _mind)
            {
                out_verts.  push_back( in_verts[id] );
                out_normals.push_back( normal       );
            }
        }
        done[id] = true;
    }
}

// -----------------------------------------------------------------------------

void Poisson_disk_sampling::sample(std::vector<Vec3_cu>& out_verts,
                          std::vector<Vec3_cu>& out_normals) const
{
    // The goal here is to build from the cluster of vertices bound to a single
    // bone of id '_bone_id' its associated sub mesh, and then sample the
    // surface of this sub mesh with the poisson disk strategy
    std::vector<Vec3_cu> in_verts;
    std::vector<int>     in_vert_ids;
    std::vector<Vec3_cu> in_normals;
    factor_samples(in_vert_ids, in_verts, in_normals);
    clamp_samples (in_vert_ids, in_verts, in_normals);

    if( in_verts.size() == 0) return;

    assert( in_vert_ids.size() == in_verts.size());
    const int size_sub_mesh = in_vert_ids.size();

    std::map<int, int> meshToCluster; // map[mesh_idx] = idx_in_verts_ids
    for(int i = 0; i < size_sub_mesh; i++)
        meshToCluster[ in_vert_ids[i] ] = i;

    // The piece of mesh defined by the bone cluster
    std::vector<int> sub_tris;
    sub_tris.reserve( size_sub_mesh * 3 * 3);

    // Building 'sub_verts' and sub_tris arrays
    std::vector<bool> done(size_sub_mesh, false);
    // Look up vertex cluster
    for(int i = 0; i < size_sub_mesh; i++)
    {
        const int idx = in_vert_ids[i];
        // Look up neighboors
        int nb_neigh = mesh->get_edge_offset(idx*2 + 1);
        int dep      = mesh->get_edge_offset(idx*2    );
        int end      = dep + nb_neigh;
        for(int n = dep; n < end; n++)
        {
            int neigh0 = mesh->get_edge( n );
            int neigh1 = mesh->get_edge((n+1) >= end ? dep : n+1);

            std::map<int, int>::iterator it0 = meshToCluster.find( neigh0 );
            std::map<int, int>::iterator it1 = meshToCluster.find( neigh1 );

            // Must be in the map (otherwise doesn't belong to the cluster)
            if(it0 != meshToCluster.end() && it1 != meshToCluster.end() )
            {
                // Must not be already treated
                if( !done[it0->second] && !done[it1->second] )
                {
                    // Add the triangles
                    sub_tris.push_back( it0->second );
                    sub_tris.push_back( it1->second );
                    sub_tris.push_back( i           );
                }
            }
        }
        // Tag vertex as treated
        done[i] = true;
    }

    // Compute the poisson disk distribution on the sub mesh
    if(sub_tris.size() > 0)
        Utils_sampling::poisson_disk(_mind, _nb_samples, in_verts, in_normals, sub_tris, out_verts, out_normals);
    std::cout << "Poisson disk sampling done. " << out_verts.size();
    std::cout << "samples created" << std::endl;
}
