#include "hip/hip_runtime.h"
/*
 Implicit skinning
 Copyright (C) 2013 Rodolphe Vaillant, Loic Barthe, Florian Cannezin,
 Gael Guennebaud, Marie Paule Cani, Damien Rohmer, Brian Wyvill,
 Olivier Gourmel

 This program is free software: you can redistribute it and/or modify
 it under the terms of the GNU General Public License 3 as published by
 the Free Software Foundation.

 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.

 You should have received a copy of the GNU General Public License
 along with this program. If not, see <http://www.gnu.org/licenses/>
 */
#include "skeleton.hpp"

#include <iostream>
#include <fstream>

#include "std_utils.hpp"
#include "blending_env.hpp"
#include "skeleton_env.hpp"
#include "precomputed_prim_env.hpp"
#include "hrbf_kernels.hpp"
#include "hrbf_env.hpp"
#include "conversions.hpp"
#include "std_utils.hpp"
#include "class_saver.hpp"
#include "loader_skel.hpp"
#include "globals.hpp"
#include "cuda_utils.hpp"

// -----------------------------------------------------------------------------

#ifndef PI
#define PI (3.14159265358979323846f)
#endif

using namespace Cuda_utils;

const float default_bone_radius = 1.f;

struct SkeletonImpl
{
    /// List of transformations associated to each bone in order to deform a mesh.
    /// A point will follow rigidly the ith bone movements if it is transformed
    /// by bone_transformations[parents[ith]].
    Cuda_utils::Host::PL_Array<Transfo> _h_transfos;
    /// same as h_transform but in device memory
    Cuda_utils::Device::Array<Transfo> _d_transfos;

    // TODO: this list might not be really needed as blending env already stores it
    /// shape of the controller associated to each joint
    /// for the gradient blending operators
    Cuda_utils::Host::Array<IBL::Ctrl_setup> _controllers;

    void init(int nb_joints)
    {
        _h_transfos.malloc(nb_joints);
        _d_transfos.malloc(nb_joints);
        _controllers.malloc(nb_joints);
    }

    typedef Cuda_utils::Host::PL_Array<Transfo> HPLA_tr;

    /// transform implicit surfaces computed with HRBF.
    /// @param global_transfos array of global transformations for each bone
    /// (device memory)
    void transform_hrbf(Skeleton *self, const Cuda_utils::Device::Array<Transfo>& d_global_transfos);

    /// transform implicit surfaces pre computed in 3D grids
    /// @param global_transfos array of global transformations for each bone
    void transform_precomputed_prim(Skeleton *self, const HPLA_tr& global_transfos);

    /// Tool function for update_vertices() method. This updates '_anim_bones'
    /// and '_anim_frames'
    void subupdate_vertices(Skeleton *self, int root, const HPLA_tr& global_transfos);
};

Transfo Skeleton::bone_anim_frame(int bone) const { return impl->_h_transfos[bone] * _bones[bone].get_frame(); }

const Transfo* Skeleton::d_transfos() const { return impl->_d_transfos.ptr(); }

void Skeleton::init(int nb_joints)
{
    impl->init(nb_joints);
    _nb_joints = nb_joints;
    _children.resize(nb_joints);
    _parents.resize(nb_joints);
    _frames.resize(nb_joints);
    _anim_frames.resize(nb_joints);
    _joints_data.resize(nb_joints);
    _anim_bones.resize(nb_joints);
    _bones.resize(nb_joints);
    _hrbf_radius.resize(nb_joints, 1.f);

    for(int i = 0; i < nb_joints; i++)
    {
        _anim_bones[i] = new Bone_ssd();

        Skeleton_env::Joint_data d;
        d._blend_type     = EJoint::MAX;
        d._ctrl_id        = Blending_env::new_ctrl_instance();
        d._bulge_strength = 0.7f;

        impl->_controllers[i] = IBL::Shape::caml();
        _joints_data[i] = d;
        Blending_env::update_controller(d._ctrl_id, impl->_controllers[i]);

        _anim_bones[i]->set_radius(default_bone_radius);
        _anim_bones[i]->_bone_id = i;

        impl->_h_transfos[i] = Transfo::identity();
    }

    _scale = 1.f;
    _offset = Vec3_cu::zero();
}

// -----------------------------------------------------------------------------

void Skeleton::init_skel_env()
{
    _skel_id = Skeleton_env::new_skel_instance(_root, _anim_bones, _parents);
    update_bones_pose();
    Skeleton_env::update_joints_data(_skel_id, _joints_data);
    Skeleton_env::update_bones_data (_skel_id, _anim_bones );
}

Skeleton::Skeleton(const Loader::Abs_skeleton& skel):
    _root(skel._root),
    impl(new SkeletonImpl())
{
    init( skel._bones.size() );

    for(unsigned i = 0; i < skel._bones.size(); i++ )
    {
        _frames    [i] = skel._bones[i]._frame;
        _parents   [i] = skel._parents[i];

        _anim_bones[i]->set_length( skel._bones[i]._length );

    }
    _children = skel._sons;

    fill_bones();
    // must be called last
    init_skel_env();
}

// -----------------------------------------------------------------------------

Skeleton::~Skeleton()
{
    for(unsigned i = 0; i < _anim_bones.size(); i++){
        _children[i].clear();
        delete _anim_bones[i];
        const int ctrl_id = _joints_data[i]._ctrl_id;
        if( ctrl_id >= 0)
            Blending_env::delete_ctrl_instance(ctrl_id);
    }

    Skeleton_env::delete_skel_instance( _skel_id );
}

void Skeleton::rec_to_string(int id, int depth, std::string& str)
{
    for (int i = 0; i < depth; ++i)
        str += "    ";

    str += "Bone: " + Std_utils::to_string(id) + " ";
    str += EBone::type_to_string( bone_type(id) ) + "\n";

    for(unsigned i = 0; i < _children[id].size(); ++i)
        rec_to_string( _children[id][i], depth+1, str);
}

// -----------------------------------------------------------------------------

std::string Skeleton::to_string()
{
    std::string str;
    rec_to_string(root(), 0, str);
    return str;
}

// -----------------------------------------------------------------------------

void Skeleton::set_joint_controller(Blending_env::Ctrl_id i,
                                    const IBL::Ctrl_setup& shape)
{
    assert( i >= 0);
    assert( i < _nb_joints);

    impl->_controllers[i] = shape;
    Blending_env::update_controller(_joints_data[i]._ctrl_id, shape);
}

// -----------------------------------------------------------------------------

void Skeleton::set_joint_blending(int i, EJoint::Joint_t type)
{
    assert( i >= 0);
    assert( i < _nb_joints);

    _joints_data[i]._blend_type = type;
    Skeleton_env::update_joints_data(_skel_id, _joints_data);
}

// -----------------------------------------------------------------------------

void Skeleton::set_joint_bulge_mag(int i, float m)
{
    assert( i >= 0);
    assert( i < _nb_joints);

    _joints_data[i]._bulge_strength = std::min(std::max(m, 0.f), 1.f);
    Skeleton_env::update_joints_data(_skel_id, _joints_data);
}

// -----------------------------------------------------------------------------

void Skeleton::set_bone(int i, Bone* b)
{
    assert(i < _nb_joints);
    assert(i >= 0);

    b->_bone_id = i;

    delete _anim_bones[i];
    _anim_bones[i] = b;

//    // TODO: to be deleted update_hrbf_id_to_bone_id();
}

// -----------------------------------------------------------------------------

void Skeleton::set_bone_radius(int i, float radius)
{
    _anim_bones[i]->set_radius(radius);
}

// -----------------------------------------------------------------------------

IBL::Ctrl_setup Skeleton::get_joint_controller(int i)
{
    assert( i >= 0);
    assert( i < _nb_joints);
    return impl->_controllers[i];
}

// -----------------------------------------------------------------------------

void Skeleton::set_bone_hrbf_radius(int i, float radius)
{
    _hrbf_radius[i] = radius;

    if(bone_type(i) == EBone::HRBF)
    {
        ((Bone_hrbf*)_anim_bones[i])->set_hrbf_radius(radius);
    }
}

// -----------------------------------------------------------------------------

int Skeleton::get_hrbf_id(Bone::Id bone_id) const
{
    assert(bone_id >= 0);
    assert(bone_id < _nb_joints);
    if(bone_type(bone_id) == EBone::HRBF)
        return ((const Bone_hrbf*)_anim_bones[bone_id])->get_hrbf().get_id();
    else
        return -1;
}

// -----------------------------------------------------------------------------

float Skeleton::get_hrbf_radius(Bone::Id bone_id) const
{
    return _hrbf_radius[bone_id];
}

Vec3_cu Skeleton::joint_pos(int joint) const {
    assert(joint >= 0        );
    assert(joint <  _nb_joints);
    return _anim_frames[joint].get_translation();
}

void SkeletonImpl::transform_hrbf(Skeleton *self, const Cuda_utils::Device::Array<Transfo>& d_global_transfos)
{
    for (int i = 0; i < self->nb_joints(); ++i)
    {
        const int id = self->get_hrbf_id(i);
        if( id > -1) HRBF_env::set_transfo(id, _h_transfos[i]);
    }

    HRBF_env::apply_hrbf_transfos();
}

// -----------------------------------------------------------------------------

void SkeletonImpl::transform_precomputed_prim(Skeleton *self, const HPLA_tr &global_transfos )
{
    // XXX: why does _nb_joints exist isntead of just _anim_bones.size()
    for( int i = 0; i < self->_nb_joints; i++)
    {
        if(self->bone_type(i) != EBone::PRECOMPUTED)
            continue;

        Bone_precomputed *bone = (Bone_precomputed*) self->_anim_bones[i];
        Precomputed_prim &prim = bone->get_primitive();
        Precomputed_env::set_transform(prim.get_id(), global_transfos[i]);
    }

    Precomputed_env::update_device_transformations();
}

void Skeleton::set_transforms(const std::vector<Transfo> &transfos)
{
    impl->_h_transfos.copy_from(transfos);
    update_bones_pose();
}

void Skeleton::update_bones_pose()
{
    // Update joints position in animated position and the associated
    // transformations
    impl->subupdate_vertices( this, _root, impl->_h_transfos );

    // Update joint positions in texture.
    impl->_d_transfos.copy_from( impl->_h_transfos );

    impl->transform_hrbf( this, impl->_d_transfos );
    impl->transform_precomputed_prim( this, impl->_h_transfos );

    // In order to this call to take effect correctly it MUST be done after
    // transform_hrbf() and transform_precomputed_prim() otherwise bones
    // positions will not be updated correctly within the Skeleton_env.
    Skeleton_env::update_bones_data(_skel_id, _anim_bones);
}

// -----------------------------------------------------------------------------

void SkeletonImpl::subupdate_vertices(Skeleton *self,  int root, const HPLA_tr& global_transfos)
{
    const Transfo tr = global_transfos[root];
    self->_anim_frames[root] = tr * self->_frames[root];

    Bone_cu b = self->_bones[root];
    self->_anim_bones[root]->set_length( b.length() );
    self->_anim_bones[root]->set_orientation(tr * b.org(), tr * b.dir());

    for(unsigned i = 0; i < self->_children[root].size(); i++)
        subupdate_vertices(self, self->_children[root][i], global_transfos);
}

// -----------------------------------------------------------------------------

/*
  // TODO: to be deleted
void Skeleton::update_hrbf_id_to_bone_id()
{
    int res = 0;
    for(int i = 0; i < _nb_joints; i++){
        if(bone_type(i) == EBone::HRBF){
            int hrbf_id = ((Bone_hrbf*)_anim_bones[i])->get_hrbf().get_id();
            res = std::max(hrbf_id , res);
        }
    }

    _hrbf_id_to_bone_id.clear();
    _hrbf_id_to_bone_id.resize(res+1);

    for(int i = 0; i < _nb_joints; i++){
        if(bone_type(i) == EBone::HRBF){
            int hrbf_id = ((Bone_hrbf*)_anim_bones[i])->get_hrbf().get_id();
            _hrbf_id_to_bone_id[hrbf_id] = i;
        }
    }
}
*/

void Skeleton::fill_bones()
{
    for(int bid = 0; bid < _nb_joints; bid++)
    {
        Vec3_cu org = _frames[bid].get_translation();
        Vec3_cu end = Vec3_cu::zero();
        int nb_sons = _children[bid].size();
        for(int s = 0; s < nb_sons; s++)
        {
            int sid = _children[bid][s];
            end += _frames[sid].get_translation();
        }
        end /= (float)nb_sons;

        if(nb_sons == 0 ){
            // We set a minimal length for the leaves
            _bones[bid] = Bone_cu(org.to_point(), _frames[bid].x(), 0.01f, 0.f);
            _anim_bones[bid]->set_length( 0.01f );
        }else{
            _bones[bid] = Bone_cu(org.to_point(), end.to_point(), 0.f);
            _anim_bones[bid]->set_length( (org-end).norm() );
        }

    }
}

// -----------------------------------------------------------------------------

Skeleton_env::DBone_id Skeleton::get_bone_didx(Bone::Id i) const {
    return Skeleton_env::bone_hidx_to_didx(_skel_id, i);
}

const Transfo&  Skeleton::get_transfo(Bone::Id bone_id) const {
    assert(bone_id >= 0);
    assert(bone_id < _nb_joints);
    return impl->_h_transfos[bone_id];
}
