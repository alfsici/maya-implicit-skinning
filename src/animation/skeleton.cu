#include "hip/hip_runtime.h"
/*
 Implicit skinning
 Copyright (C) 2013 Rodolphe Vaillant, Loic Barthe, Florian Cannezin,
 Gael Guennebaud, Marie Paule Cani, Damien Rohmer, Brian Wyvill,
 Olivier Gourmel

 This program is free software: you can redistribute it and/or modify
 it under the terms of the GNU General Public License 3 as published by
 the Free Software Foundation.

 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.

 You should have received a copy of the GNU General Public License
 along with this program. If not, see <http://www.gnu.org/licenses/>
 */
#include "skeleton.hpp"

#include <iostream>
#include <fstream>

#include "std_utils.hpp"
#include "color.hpp"
#include "blending_env.hpp"
#include "skeleton_env.hpp"
#include "hrbf_kernels.hpp"
#include "conversions.hpp"
#include "std_utils.hpp"
#include "class_saver.hpp"
#include "graph.hpp"
#include "loader_skel.hpp"
#include "globals.hpp"

// -----------------------------------------------------------------------------

#ifndef PI
#define PI (3.14159265358979323846f)
#endif

using namespace Cuda_utils;

const float default_bone_radius = 1.f;

// -----------------------------------------------------------------------------

/// @param g : graph to look up
/// @param curr : current node of 'g'
/// @param prev : previous (i.e parent) node of g(curr)
/// @param res : newly created tree with nodes duplicated when several sons
/// @param pt : current parent in res
static void rec_gen_bone_graph(const Graph& g,
                               int curr,
                               int prev,
                               Graph& res,
                               int pt)
{
    // Current is a leaf:
    if( g._neighs[curr].size() < 2  && prev != -1)
    {
        int id = res.push_vertex( g._vertices[curr] );
        if(pt != -1) res.push_edge( Graph::Edge(pt, id) );
        return;
    }

    for(unsigned i = 0; i < g._neighs[curr].size(); ++i)
    {
        const int arc = g._neighs[curr][i];

        if(arc == prev) continue; // skip parent

        int id = res.push_vertex( g._vertices[curr] );
        if(pt != -1) res.push_edge( Graph::Edge(pt, id) );

        rec_gen_bone_graph(g, arc, curr, res, id);
    }

}

// -----------------------------------------------------------------------------

static Graph gen_bone_graph(const Graph& g, int root)
{
    Graph res( g._offset, g._scale);
    if( g._neighs[root].size() > 1)
    {
        // When root has more than 1 son it will be duplicated, hence no root
        // node will exists. We need to create it
        Vec3_cu v = g._vertices[root];
        v.x += 0.1f; // offset so that it has not zero length
        int id = res.push_vertex( v );
        rec_gen_bone_graph(g, root, -1, res, id);

    }
    else if ( g._neighs[root].size() == 0)
        res.push_vertex( g._vertices[root] );
    else
        rec_gen_bone_graph(g, root, -1, res, -1);

    return  res;
}

// -----------------------------------------------------------------------------

void Skeleton::init(int nb_joints)
{
    _nb_joints = nb_joints;
    _children.resize(nb_joints);
    _parents.resize(nb_joints);
    _frames.resize(nb_joints);
    _lcl_frames.resize(nb_joints);
    _anim_frames.resize(nb_joints);
    _saved_transfos.resize(nb_joints);
    _h_transfos.malloc(nb_joints);
    _d_transfos.malloc(nb_joints);
    _joints_data.resize(nb_joints);
    _controllers.malloc(nb_joints);
    _anim_bones.resize(nb_joints);
    _bones.resize(nb_joints);
    _hrbf_radius.resize(nb_joints, 1.f);

    for(int i = 0; i < nb_joints; i++)
    {
        _anim_bones[i] = new Bone_ssd();

        Skeleton_env::Joint_data d;
        d._blend_type     = EJoint::MAX;
        d._ctrl_id        = Blending_env::new_ctrl_instance();
        d._bulge_strength = 0.7f;

        _controllers[i] = IBL::Shape::caml();
        _joints_data[i] = d;
        Blending_env::update_controller(d._ctrl_id, _controllers[i]);

        _anim_bones[i]->set_radius(default_bone_radius);
        _anim_bones[i]->_bone_id = i;

        _h_transfos[i] = Transfo::identity();
    }

    _scale = 1.f;
    _offset = Vec3_cu::zero();
}

// -----------------------------------------------------------------------------

void Skeleton::init_skel_env()
{
    _skel_id = Skeleton_env::new_skel_instance(_root, _anim_bones, _parents);
    Skeleton_env::update_joints_data(_skel_id, _joints_data);
    Skeleton_env::update_bones_data (_skel_id, _anim_bones );
}

// -----------------------------------------------------------------------------

Skeleton::Skeleton(const Graph& graph, int root)
{
    assert( !graph.is_cycles(root) );

    // from the graph we generate another graph were nodes with more than one
    // son are duplicated to obtain as many nodes as sons. The new nodes keeps
    // the position and arcs of the old node.
    // Later each node will give rise to a single bone.
    Graph g = gen_bone_graph(graph, root);
    // The generated graph has always root set to node 0
    _root = root = 0;

    init( g.nb_vertices() );

    _parents[_root] = -1;
    Mat3_cu id = Mat3_cu::identity();
    _frames     [_root] = Transfo( id, g._vertices[_root] );
    _lcl_frames [_root] = _frames[_root].fast_invert();

    Graph tmp = g;
    fill_children( tmp, _root );
    fill_frames( g );
    fill_bones();
    // must be called last
    init_skel_env();
}

// -----------------------------------------------------------------------------

Skeleton::Skeleton(const Loader::Abs_skeleton& skel) : _root(skel._root)
{
    init( skel._bones.size() );

    for(unsigned i = 0; i < skel._bones.size(); i++ )
    {
        Transfo tr = skel._bones[i]._frame;
        _frames    [i] = tr;
        _lcl_frames[i] = tr.fast_invert();
        _parents   [i] = skel._parents[i];

        _anim_bones[i]->set_length( skel._bones[i]._length );

    }
    _children = skel._sons;

    fill_bones();
    // must be called last
    init_skel_env();
}

// -----------------------------------------------------------------------------

Skeleton::~Skeleton()
{
    for(unsigned i = 0; i < _anim_bones.size(); i++){
        _children[i].clear();
        delete _anim_bones[i];
        const int ctrl_id = _joints_data[i]._ctrl_id;
        if( ctrl_id >= 0)
            Blending_env::delete_ctrl_instance(ctrl_id);
    }

    Skeleton_env::delete_skel_instance( _skel_id );
}

// -----------------------------------------------------------------------------

void Skeleton::reset()
{
    for(int i = 0; i < nb_joints(); i++){
        _saved_transfos[i] = _h_transfos[i];
        _h_transfos[i] = Transfo::identity();
    }
    update_bones_pose();
}

// -----------------------------------------------------------------------------

void Skeleton::unreset()
{
    for(int i = 0; i < nb_joints(); i++)
        _h_transfos[i] = _saved_transfos[i];
    update_bones_pose();
}

// -----------------------------------------------------------------------------

void Skeleton::compute_joints_half_angles(HA_Vec3_cu& half_angles,
                                          HA_Vec3_cu& orthos)
{
    for(int i = 0; i < nb_joints(); i++)
    {
        if(i == _root  || is_leaf(i))
        {
            half_angles[i] = Vec3_cu(0.f, 0.f, 0.f);
            orthos     [i] = Vec3_cu(0.f, 0.f, 0.f);
        }
        else
        {
            Vec3_cu null  = Vec3_cu::zero();
            Vec3_cu half  = Vec3_cu::zero();
            Vec3_cu ortho = Vec3_cu::zero();
            Vec3_cu v0    = joint_pos(_parents[i]) - joint_pos(i);

            const std::vector<int>& sons = get_sons(i);
            for(unsigned p = 0; p < sons.size(); ++p)
            {
                Vec3_cu v1 = joint_pos(sons[p]) - joint_pos(i);

                Vec3_cu temp = v0.cross(v1);
                if(temp.norm() >= 0.0001f)
                {
                    half  = half  + (v0 + v1);
                    ortho = ortho + temp;
                }
            }

            half  = half  * (1.f / (float)sons.size());
            ortho = ortho * (1.f / (float)sons.size());

            half_angles[i] = half. norm() > 0.0001f ? half. normalized() : null;
            orthos     [i] = ortho.norm() > 0.0001f ? ortho.normalized() : null;
        }
    }
}

// -----------------------------------------------------------------------------

void Skeleton::rec_to_string(int id, int depth, std::string& str)
{
    for (int i = 0; i < depth; ++i)
        str += "    ";

    str += "Bone: " + Std_utils::to_string(id) + " ";
    str += EBone::type_to_string( bone_type(id) ) + "\n";

    for(unsigned i = 0; i < _children[id].size(); ++i)
        rec_to_string( _children[id][i], depth+1, str);
}

// -----------------------------------------------------------------------------

std::string Skeleton::to_string()
{
    std::string str;
    rec_to_string(root(), 0, str);
    return str;
}

// -----------------------------------------------------------------------------

void Skeleton::set_joint_controller(Blending_env::Ctrl_id i,
                                    const IBL::Ctrl_setup& shape)
{
    assert( i >= 0);
    assert( i < _nb_joints);

    _controllers[i] = shape;
    Blending_env::update_controller(_joints_data[i]._ctrl_id, shape);
}

// -----------------------------------------------------------------------------

void Skeleton::set_joint_blending(int i, EJoint::Joint_t type)
{
    assert( i >= 0);
    assert( i < _nb_joints);

    _joints_data[i]._blend_type = type;
    Skeleton_env::update_joints_data(_skel_id, _joints_data);
}

// -----------------------------------------------------------------------------

void Skeleton::set_joint_bulge_mag(int i, float m)
{
    assert( i >= 0);
    assert( i < _nb_joints);

    _joints_data[i]._bulge_strength = std::min(std::max(m, 0.f), 1.f);
    Skeleton_env::update_joints_data(_skel_id, _joints_data);
}

// -----------------------------------------------------------------------------

void Skeleton::set_bone(int i, Bone* b)
{
    assert(i < _nb_joints);
    assert(i >= 0);

    b->_bone_id = i;

    delete _anim_bones[i];
    _anim_bones[i] = b;

//    // TODO: to be deleted update_hrbf_id_to_bone_id();
}

// -----------------------------------------------------------------------------

void Skeleton::set_bone_radius(int i, float radius)
{
    _anim_bones[i]->set_radius(radius);
}

// -----------------------------------------------------------------------------

IBL::Ctrl_setup Skeleton::get_joint_controller(int i)
{
    assert( i >= 0);
    assert( i < _nb_joints);
    return _controllers[i];
}

// -----------------------------------------------------------------------------

void Skeleton::set_bone_hrbf_radius(int i, float radius)
{
    _hrbf_radius[i] = radius;

    if(bone_type(i) == EBone::HRBF)
    {
        ((Bone_hrbf*)_anim_bones[i])->set_hrbf_radius(radius);
    }
}

// -----------------------------------------------------------------------------

int Skeleton::get_hrbf_id(Bone::Id bone_id) const
{
    assert(bone_id >= 0);
    assert(bone_id < _nb_joints);
    if(bone_type(bone_id) == EBone::HRBF)
        return ((const Bone_hrbf*)_anim_bones[bone_id])->get_hrbf().get_id();
    else
        return -1;
}

// -----------------------------------------------------------------------------

float Skeleton::get_hrbf_radius(Bone::Id bone_id)
{
    return _hrbf_radius[bone_id];
}

// -----------------------------------------------------------------------------

void Skeleton::set_joint_rest_pos(int joint_id, const Point_cu& pt)
{
    _frames[joint_id].set_translation( pt.to_vector() );
    _lcl_frames[joint_id] = _frames[joint_id].fast_invert();
    fill_bones();
}

// -----------------------------------------------------------------------------

void Skeleton::set_offset_scale(const Vec3_cu& offset, float scale)
{
    _offset = offset;
    _scale  = scale;
    for(int i = 0; i < _nb_joints; i++ )
    {
        Transfo tr = Transfo::scale(scale) * Transfo::translate(offset) * _frames[i];
        _frames[i] = tr;
        _lcl_frames[i] = tr.fast_invert();
        _anim_bones[i]->set_length( _anim_bones[i]->length() * scale );
    }
    fill_bones();
}

// -----------------------------------------------------------------------------

Vec3_cu Skeleton::joint_pos(int joint) const {
    assert(joint >= 0        );
    assert(joint <  _nb_joints);
    return _anim_frames[joint].get_translation();
}

// -----------------------------------------------------------------------------

Vec3_cu Skeleton::joint_rest_pos(int joint){
    assert(joint >= 0        );
    assert(joint <  _nb_joints);
    return _frames[joint].get_translation();
}

// -----------------------------------------------------------------------------

void Skeleton::transform_hrbf(const Cuda_utils::Device::Array<Transfo>& d_global_transfos)
{
    for (int i = 0; i < nb_joints(); ++i)
    {
        const int id = get_hrbf_id(i);
        if( id > -1) HRBF_env::set_transfo(id, _h_transfos[i]);
    }

    HRBF_env::apply_hrbf_transfos();
}

// -----------------------------------------------------------------------------

void Skeleton::transform_precomputed_prim(const HPLA_tr &global_transfos )
{

    for( int i = 0; i < _nb_joints; i++)
    {
        if(bone_type(i) != EBone::PRECOMPUTED)
            continue;

        Bone_precomputed *bone = (Bone_precomputed*) _anim_bones[i];
        Precomputed_prim &prim = bone->get_primitive();
        Precomputed_env::set_transform(prim.get_id(), global_transfos[i]);
    }

    Precomputed_env::update_device_transformations();
}

void Skeleton::set_transforms(const std::vector<Transfo> &transfos)
{
    _h_transfos.copy_from(transfos);
    update_bones_pose();
}

void Skeleton::update_bones_pose()
{
    // Update joints position in animated position and the associated
    // transformations
    subupdate_vertices( _root, _h_transfos );

    // Update joint positions in texture.
    _d_transfos.copy_from( _h_transfos );

    transform_hrbf( _d_transfos );
    transform_precomputed_prim( _h_transfos );

    // In order to this call to take effect correctly it MUST be done after
    // transform_hrbf() and transform_precomputed_prim() otherwise bones
    // positions will not be updated correctly within the Skeleton_env.
    Skeleton_env::update_bones_data(_skel_id, _anim_bones);
}

// -----------------------------------------------------------------------------

void Skeleton::subupdate_vertices( int root,
                                   const HPLA_tr& global_transfos)
{
    const Transfo tr = global_transfos[root];
    _anim_frames[root] = tr * _frames[root];

    Bone_cu b = _bones[root];
    _anim_bones[root]->set_length( b.length() );
    _anim_bones[root]->set_orientation(tr * b.org(), tr * b.dir());

    for(unsigned i = 0; i < _children[root].size(); i++)
        subupdate_vertices(_children[root][i], global_transfos);
}

// -----------------------------------------------------------------------------

/*
  // TODO: to be deleted
void Skeleton::update_hrbf_id_to_bone_id()
{
    int res = 0;
    for(int i = 0; i < _nb_joints; i++){
        if(bone_type(i) == EBone::HRBF){
            int hrbf_id = ((Bone_hrbf*)_anim_bones[i])->get_hrbf().get_id();
            res = std::max(hrbf_id , res);
        }
    }

    _hrbf_id_to_bone_id.clear();
    _hrbf_id_to_bone_id.resize(res+1);

    for(int i = 0; i < _nb_joints; i++){
        if(bone_type(i) == EBone::HRBF){
            int hrbf_id = ((Bone_hrbf*)_anim_bones[i])->get_hrbf().get_id();
            _hrbf_id_to_bone_id[hrbf_id] = i;
        }
    }
}
*/

// -----------------------------------------------------------------------------

void Skeleton::fill_children(Graph& g, int root)
{
    std::vector<int> to_pop;
    to_pop.reserve( 2 * g.nb_edges() );
    for(int i = 0; i < (int)g.nb_edges(); i++)
    {
        const Graph::Edge& e = g._edges[i];
        if(e.a == root){
            to_pop.push_back(e.b);
            _children[root].push_back(e.b);
            _parents[e.b] = root;
            Std_utils::pop(g._edges, i);
            i = -1;
        } else {
            if(e.b == root){
                to_pop.push_back(e.a);
                _children[root].push_back(e.a);
                _parents[e.a] = root;
                Std_utils::pop(g._edges, i);
                i = -1;
            }
        }
    }

    for(unsigned i = 0; i < to_pop.size(); i++)
        fill_children(g, to_pop[i]);
}

// -----------------------------------------------------------------------------

void Skeleton::fill_frames(const Graph& g)
{
    for(int i = 0; i < _nb_joints; i++)
    {
        Vec3_cu org = g.get_vertex( i );
        Vec3_cu end = Vec3_cu::zero();
        int nb_sons = _children[i].size();
        for(int s = 0; s < nb_sons; s++)
        {
            int sid = _children[i][s];
            end += g.get_vertex( sid );
        }

        Vec3_cu x, y, z;
        if( nb_sons > 0){
            end /= (float)nb_sons;
            x = (end - org).normalized();
        }else
            x = Vec3_cu::unit_x();

        x.coordinate_system(y, z);

        Transfo tr(Mat3_cu(x, y, z), org);
        _frames    [i] = tr;
        _lcl_frames[i] = tr.fast_invert();
    }
}

// -----------------------------------------------------------------------------

void Skeleton::fill_bones()
{
    for(int bid = 0; bid < _nb_joints; bid++)
    {
        Vec3_cu org = _frames[bid].get_translation();
        Vec3_cu end = Vec3_cu::zero();
        int nb_sons = _children[bid].size();
        for(int s = 0; s < nb_sons; s++)
        {
            int sid = _children[bid][s];
            end += _frames[sid].get_translation();
        }
        end /= (float)nb_sons;

        if(nb_sons == 0 ){
            // We set a minimal length for the leaves
            _bones[bid] = Bone_cu(org.to_point(), _frames[bid].x(), 0.01f, 0.f);
            _anim_bones[bid]->set_length( 0.01f );
        }else{
            _bones[bid] = Bone_cu(org.to_point(), end.to_point(), 0.f);
            _anim_bones[bid]->set_length( (org-end).norm() );
        }

    }
}

// -----------------------------------------------------------------------------

Skeleton_env::DBone_id Skeleton::get_bone_didx(Bone::Id i) const {
    return Skeleton_env::bone_hidx_to_didx(_skel_id, i);
}

// -----------------------------------------------------------------------------

int Skeleton::get_nb_bone_of_type(EBone::Bone_t type)
{
    int acc = 0;
    for(int i = 0; i < _nb_joints; i++)
        if(bone_type(i) == type)
            acc++;

    return acc;
}

// -----------------------------------------------------------------------------

const Transfo&  Skeleton::get_transfo(Bone::Id bone_id) const {
    assert(bone_id >= 0);
    assert(bone_id < _nb_joints);
    return _h_transfos[bone_id];
}
