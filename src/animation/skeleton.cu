#include "hip/hip_runtime.h"
/*
 Implicit skinning
 Copyright (C) 2013 Rodolphe Vaillant, Loic Barthe, Florian Cannezin,
 Gael Guennebaud, Marie Paule Cani, Damien Rohmer, Brian Wyvill,
 Olivier Gourmel

 This program is free software: you can redistribute it and/or modify
 it under the terms of the GNU General Public License 3 as published by
 the Free Software Foundation.

 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.

 You should have received a copy of the GNU General Public License
 along with this program. If not, see <http://www.gnu.org/licenses/>
 */
#include "skeleton.hpp"

#include <iostream>
#include <fstream>

#include "std_utils.hpp"
#include "blending_env.hpp"
#include "skeleton_env.hpp"
#include "hrbf_kernels.hpp"
#include "conversions.hpp"
#include "std_utils.hpp"
#include "class_saver.hpp"
#include "loader_skel.hpp"
#include "globals.hpp"

// -----------------------------------------------------------------------------

#ifndef PI
#define PI (3.14159265358979323846f)
#endif

using namespace Cuda_utils;

const float default_bone_radius = 1.f;

void Skeleton::init(int nb_joints)
{
    _nb_joints = nb_joints;
    _children.resize(nb_joints);
    _parents.resize(nb_joints);
    _frames.resize(nb_joints);
    _anim_frames.resize(nb_joints);
    _saved_transfos.resize(nb_joints);
    _h_transfos.malloc(nb_joints);
    _d_transfos.malloc(nb_joints);
    _joints_data.resize(nb_joints);
    _controllers.malloc(nb_joints);
    _anim_bones.resize(nb_joints);
    _bones.resize(nb_joints);
    _hrbf_radius.resize(nb_joints, 1.f);

    for(int i = 0; i < nb_joints; i++)
    {
        _anim_bones[i] = new Bone_ssd();

        Skeleton_env::Joint_data d;
        d._blend_type     = EJoint::MAX;
        d._ctrl_id        = Blending_env::new_ctrl_instance();
        d._bulge_strength = 0.7f;

        _controllers[i] = IBL::Shape::caml();
        _joints_data[i] = d;
        Blending_env::update_controller(d._ctrl_id, _controllers[i]);

        _anim_bones[i]->set_radius(default_bone_radius);
        _anim_bones[i]->_bone_id = i;

        _h_transfos[i] = Transfo::identity();
    }

    _scale = 1.f;
    _offset = Vec3_cu::zero();
}

// -----------------------------------------------------------------------------

void Skeleton::init_skel_env()
{
    _skel_id = Skeleton_env::new_skel_instance(_root, _anim_bones, _parents);
    update_bones_pose();
    Skeleton_env::update_joints_data(_skel_id, _joints_data);
    Skeleton_env::update_bones_data (_skel_id, _anim_bones );
}

Skeleton::Skeleton(const Loader::Abs_skeleton& skel) : _root(skel._root)
{
    init( skel._bones.size() );

    for(unsigned i = 0; i < skel._bones.size(); i++ )
    {
        _frames    [i] = skel._bones[i]._frame;
        _parents   [i] = skel._parents[i];

        _anim_bones[i]->set_length( skel._bones[i]._length );

    }
    _children = skel._sons;

    fill_bones();
    // must be called last
    init_skel_env();
}

// -----------------------------------------------------------------------------

Skeleton::~Skeleton()
{
    for(unsigned i = 0; i < _anim_bones.size(); i++){
        _children[i].clear();
        delete _anim_bones[i];
        const int ctrl_id = _joints_data[i]._ctrl_id;
        if( ctrl_id >= 0)
            Blending_env::delete_ctrl_instance(ctrl_id);
    }

    Skeleton_env::delete_skel_instance( _skel_id );
}

// -----------------------------------------------------------------------------

void Skeleton::reset()
{
    for(int i = 0; i < nb_joints(); i++){
        _saved_transfos[i] = _h_transfos[i];
        _h_transfos[i] = Transfo::identity();
    }
    update_bones_pose();
}

// -----------------------------------------------------------------------------

void Skeleton::unreset()
{
    for(int i = 0; i < nb_joints(); i++)
        _h_transfos[i] = _saved_transfos[i];
    update_bones_pose();
}

void Skeleton::rec_to_string(int id, int depth, std::string& str)
{
    for (int i = 0; i < depth; ++i)
        str += "    ";

    str += "Bone: " + Std_utils::to_string(id) + " ";
    str += EBone::type_to_string( bone_type(id) ) + "\n";

    for(unsigned i = 0; i < _children[id].size(); ++i)
        rec_to_string( _children[id][i], depth+1, str);
}

// -----------------------------------------------------------------------------

std::string Skeleton::to_string()
{
    std::string str;
    rec_to_string(root(), 0, str);
    return str;
}

// -----------------------------------------------------------------------------

void Skeleton::set_joint_controller(Blending_env::Ctrl_id i,
                                    const IBL::Ctrl_setup& shape)
{
    assert( i >= 0);
    assert( i < _nb_joints);

    _controllers[i] = shape;
    Blending_env::update_controller(_joints_data[i]._ctrl_id, shape);
}

// -----------------------------------------------------------------------------

void Skeleton::set_joint_blending(int i, EJoint::Joint_t type)
{
    assert( i >= 0);
    assert( i < _nb_joints);

    _joints_data[i]._blend_type = type;
    Skeleton_env::update_joints_data(_skel_id, _joints_data);
}

// -----------------------------------------------------------------------------

void Skeleton::set_joint_bulge_mag(int i, float m)
{
    assert( i >= 0);
    assert( i < _nb_joints);

    _joints_data[i]._bulge_strength = std::min(std::max(m, 0.f), 1.f);
    Skeleton_env::update_joints_data(_skel_id, _joints_data);
}

// -----------------------------------------------------------------------------

void Skeleton::set_bone(int i, Bone* b)
{
    assert(i < _nb_joints);
    assert(i >= 0);

    b->_bone_id = i;

    delete _anim_bones[i];
    _anim_bones[i] = b;

//    // TODO: to be deleted update_hrbf_id_to_bone_id();
}

// -----------------------------------------------------------------------------

void Skeleton::set_bone_radius(int i, float radius)
{
    _anim_bones[i]->set_radius(radius);
}

// -----------------------------------------------------------------------------

IBL::Ctrl_setup Skeleton::get_joint_controller(int i)
{
    assert( i >= 0);
    assert( i < _nb_joints);
    return _controllers[i];
}

// -----------------------------------------------------------------------------

void Skeleton::set_bone_hrbf_radius(int i, float radius)
{
    _hrbf_radius[i] = radius;

    if(bone_type(i) == EBone::HRBF)
    {
        ((Bone_hrbf*)_anim_bones[i])->set_hrbf_radius(radius);
    }
}

// -----------------------------------------------------------------------------

int Skeleton::get_hrbf_id(Bone::Id bone_id) const
{
    assert(bone_id >= 0);
    assert(bone_id < _nb_joints);
    if(bone_type(bone_id) == EBone::HRBF)
        return ((const Bone_hrbf*)_anim_bones[bone_id])->get_hrbf().get_id();
    else
        return -1;
}

// -----------------------------------------------------------------------------

float Skeleton::get_hrbf_radius(Bone::Id bone_id)
{
    return _hrbf_radius[bone_id];
}

Vec3_cu Skeleton::joint_pos(int joint) const {
    assert(joint >= 0        );
    assert(joint <  _nb_joints);
    return _anim_frames[joint].get_translation();
}

void Skeleton::transform_hrbf(const Cuda_utils::Device::Array<Transfo>& d_global_transfos)
{
    for (int i = 0; i < nb_joints(); ++i)
    {
        const int id = get_hrbf_id(i);
        if( id > -1) HRBF_env::set_transfo(id, _h_transfos[i]);
    }

    HRBF_env::apply_hrbf_transfos();
}

// -----------------------------------------------------------------------------

void Skeleton::transform_precomputed_prim(const HPLA_tr &global_transfos )
{
    // XXX: why does _nb_joints exist isntead of just _anim_bones.size()
    for( int i = 0; i < _nb_joints; i++)
    {
        if(bone_type(i) != EBone::PRECOMPUTED)
            continue;

        Bone_precomputed *bone = (Bone_precomputed*) _anim_bones[i];
        Precomputed_prim &prim = bone->get_primitive();
        Precomputed_env::set_transform(prim.get_id(), global_transfos[i]);
    }

    Precomputed_env::update_device_transformations();
}

void Skeleton::set_transforms(const std::vector<Transfo> &transfos)
{
    _h_transfos.copy_from(transfos);
    update_bones_pose();
}

void Skeleton::update_bones_pose()
{
    // Update joints position in animated position and the associated
    // transformations
    subupdate_vertices( _root, _h_transfos );

    // Update joint positions in texture.
    _d_transfos.copy_from( _h_transfos );

    transform_hrbf( _d_transfos );
    transform_precomputed_prim( _h_transfos );

    // In order to this call to take effect correctly it MUST be done after
    // transform_hrbf() and transform_precomputed_prim() otherwise bones
    // positions will not be updated correctly within the Skeleton_env.
    Skeleton_env::update_bones_data(_skel_id, _anim_bones);
}

// -----------------------------------------------------------------------------

void Skeleton::subupdate_vertices( int root,
                                   const HPLA_tr& global_transfos)
{
    const Transfo tr = global_transfos[root];
    _anim_frames[root] = tr * _frames[root];

    Bone_cu b = _bones[root];
    _anim_bones[root]->set_length( b.length() );
    _anim_bones[root]->set_orientation(tr * b.org(), tr * b.dir());

    for(unsigned i = 0; i < _children[root].size(); i++)
        subupdate_vertices(_children[root][i], global_transfos);
}

// -----------------------------------------------------------------------------

/*
  // TODO: to be deleted
void Skeleton::update_hrbf_id_to_bone_id()
{
    int res = 0;
    for(int i = 0; i < _nb_joints; i++){
        if(bone_type(i) == EBone::HRBF){
            int hrbf_id = ((Bone_hrbf*)_anim_bones[i])->get_hrbf().get_id();
            res = std::max(hrbf_id , res);
        }
    }

    _hrbf_id_to_bone_id.clear();
    _hrbf_id_to_bone_id.resize(res+1);

    for(int i = 0; i < _nb_joints; i++){
        if(bone_type(i) == EBone::HRBF){
            int hrbf_id = ((Bone_hrbf*)_anim_bones[i])->get_hrbf().get_id();
            _hrbf_id_to_bone_id[hrbf_id] = i;
        }
    }
}
*/

void Skeleton::fill_bones()
{
    for(int bid = 0; bid < _nb_joints; bid++)
    {
        Vec3_cu org = _frames[bid].get_translation();
        Vec3_cu end = Vec3_cu::zero();
        int nb_sons = _children[bid].size();
        for(int s = 0; s < nb_sons; s++)
        {
            int sid = _children[bid][s];
            end += _frames[sid].get_translation();
        }
        end /= (float)nb_sons;

        if(nb_sons == 0 ){
            // We set a minimal length for the leaves
            _bones[bid] = Bone_cu(org.to_point(), _frames[bid].x(), 0.01f, 0.f);
            _anim_bones[bid]->set_length( 0.01f );
        }else{
            _bones[bid] = Bone_cu(org.to_point(), end.to_point(), 0.f);
            _anim_bones[bid]->set_length( (org-end).norm() );
        }

    }
}

// -----------------------------------------------------------------------------

Skeleton_env::DBone_id Skeleton::get_bone_didx(Bone::Id i) const {
    return Skeleton_env::bone_hidx_to_didx(_skel_id, i);
}

const Transfo&  Skeleton::get_transfo(Bone::Id bone_id) const {
    assert(bone_id >= 0);
    assert(bone_id < _nb_joints);
    return _h_transfos[bone_id];
}
