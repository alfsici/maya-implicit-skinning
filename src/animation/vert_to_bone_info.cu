#include "hip/hip_runtime.h"
#include "vert_to_bone_info.hpp"
#include "mesh.hpp"
#include "skeleton.hpp"


VertToBoneInfo::VertToBoneInfo(const Skeleton *skel, const Mesh *mesh, const std::vector< std::vector<Bone::Id> > &bones_per_vertex):
    bones_per_vertex(bones_per_vertex)
{
    // Create an entry for each bone, even if the value is zero.
    for(Bone::Id bone_id: skel->get_bone_ids())
        h_verts_id_per_bone[bone_id];

    for(int vert_idx = 0; vert_idx < (int) bones_per_vertex.size(); ++vert_idx)
    {
        for(int bone_idx: bones_per_vertex[vert_idx])
            h_verts_id_per_bone[bone_idx].push_back(vert_idx);
    }
}

void VertToBoneInfo::get_default_junction_radius(const Skeleton *skel, const Mesh *mesh, std::map<Bone::Id,float> &nearest_rad) const
{
    const int nb_verts  = mesh->get_nb_vertices();

    const float inf = std::numeric_limits<float>::infinity();

    // Junction radius is nearest vertex distance
    for(Bone::Id bone_id: skel->get_bone_ids())
        nearest_rad[bone_id] = inf;

    for(int i = 0; i < nb_verts; i++)
    {
        const Point_cu vert = mesh->get_vertex(i).to_point();
        const std::vector<Bone::Id> &bone_ids = bones_per_vertex.at(i);
        for(Bone::Id bone_id: bone_ids) {
            float dist = skel->get_bone(bone_id)->dist_to(vert);

            nearest_rad[bone_id] = std::min(nearest_rad[bone_id], dist);
        }
    }

    for(Bone::Id i: skel->get_bone_ids())
    {
        if(nearest_rad[i] == inf)
            nearest_rad[i] = 1.f;
    }
}

void VertToBoneInfo::get_default_hrbf_radius(const Skeleton *skel, const Mesh *mesh, std::map<Bone::Id,float> &out) const
{
    const int nb_verts  = mesh->get_nb_vertices();

    std::map<Bone::Id, float> farthest_rad;
    for(int i = 0; i < nb_verts; i++)
    {
        const Point_cu vert = mesh->get_vertex(i).to_point();
        const std::vector<Bone::Id> &bone_ids = bones_per_vertex.at(i);
        for(Bone::Id bone_id: bone_ids) {
            float dist = skel->get_bone(bone_id)->dist_to( vert );

            farthest_rad[bone_id] = std::max(farthest_rad[bone_id], dist);
        }
    }

    // HRBF compact support radius is farthest vertex distance
    for(Bone::Id i: skel->get_bone_ids())
        out[i] = farthest_rad[i] == 0.f ? 1.f : farthest_rad[i];
}
