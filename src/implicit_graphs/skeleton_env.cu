#include "hip/hip_runtime.h"
/*
 Implicit skinning
 Copyright (C) 2013 Rodolphe Vaillant, Loic Barthe, Florian Cannezin,
 Gael Guennebaud, Marie Paule Cani, Damien Rohmer, Brian Wyvill,
 Olivier Gourmel

 This program is free software: you can redistribute it and/or modify
 it under the terms of the GNU General Public License 3 as published by
 the Free Software Foundation.

 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.

 You should have received a copy of the GNU General Public License
 along with this program. If not, see <http://www.gnu.org/licenses/>
 */
#include "skeleton_env.hpp"
#include "skeleton.hpp"

#include "std_utils.hpp"
#include "grid.hpp"
#include "tree_cu.hpp"
#include "tree.hpp"
#include <list>
#include <deque>
#include <map>
#include <set>

namespace { __device__ void fix_debug() { } }

// =============================================================================
namespace Skeleton_env {
// =============================================================================

bool allocated = false;
bool binded;

// -----------------------------------------------------------------------------
/// @name GPU friendly datas
// -----------------------------------------------------------------------------

Bone_tex* hd_bone_arrays = 0;

/// Concatenated blendind list for every skeletons
Cuda_utils::HD_Array<Cluster_cu> hd_blending_list;

Cuda_utils::HD_Array<Cluster_data> hd_cluster_data;

texture<int4, 1, hipReadModeElementType> tex_blending_list;
texture<int4, 1, hipReadModeElementType> tex_grid_list;
texture<int, 1, hipReadModeElementType> tex_grid;
texture<float4, 1, hipReadModeElementType> tex_grid_bbox;
texture<int2, 1, hipReadModeElementType> tex_offset;
texture<float, 1, hipReadModeElementType> tex_bulge_strength;
texture<int, 1, hipReadModeElementType> tex_bone_type;
texture<int   , 1, hipReadModeElementType> tex_bone_hrbf;
texture<int   , 1, hipReadModeElementType> tex_bone_precomputed;

/// Concatenated blendind list for every skeletons in each grid cell not empty.
/// Each cell store a single blending list.
/**
 * @code
 *  |-list0-|-list1-|-list2-| |-list0-|-list1-|
 *  *-----------------------* *---------------*
 *            skel0                 skel1
 * @endcode
*/

class SkeletonEnv
{
public:
    SkeletonEnv();
    ~SkeletonEnv();

    /// Skeletons instance, a skeleton must be a tree with one connex componant.
    Tree *h_tree;

    /// Acceleration structure for the different skeletons in the environment

    Grid *h_grid;

    Tree_cu *h_tree_cu_instance;
};

std::deque<SkeletonEnv *> h_envs;

Cuda_utils::HD_Array<Cluster_cu> hd_grid_blending_list;

/// Concatenated datas corresponding to clusters listed hd_grid_blending_list
Cuda_utils::HD_Array<Cluster_data> hd_grid_data;
/// Table of indirection which maps grid cells to blending list.
/// hd_grid[ hd_offset[Skel_id].grid_data + cell_idx] == offset in hd_grid_blending_list or -1 if empty cell
Cuda_utils::HD_Array<int> hd_grid;

Cuda_utils::HD_Array<float4> hd_grid_bbox;

/// d_offset[Skel_id] == offset in lists or grid
Cuda_utils::HD_Array<Offset>  hd_offset; // maybe we should cut in half offset and allocate it in advance at each new skeleton instances

// -----------------------------------------------------------------------------
/// @name CPU friendly datas
// -----------------------------------------------------------------------------

/// user idx to device bone idx
std::map<Hbone_id, DBone_id> _hidx_to_didx;
/// device bone idx to user idx
std::map<DBone_id, Hbone_id> _didx_to_hidx;



SkeletonEnv::SkeletonEnv()
{
    h_tree = NULL;
    h_tree_cu_instance = NULL;
    h_grid = NULL;
}

SkeletonEnv::~SkeletonEnv()
{
    delete h_tree;
    delete h_tree_cu_instance;
    delete h_grid;
}


void bind()
{
    using namespace Cuda_utils;
    
    binded = true;
    // Initialize texture and bind them to the arrays
    if(hd_bone_arrays != 0)
    {
        hd_bone_arrays->hd_bone_hrbf.       device_array().bind_tex(tex_bone_hrbf       );
        hd_bone_arrays->hd_bone_precomputed.device_array().bind_tex(tex_bone_precomputed);
        hd_bone_arrays->hd_bone_types.      device_array().bind_tex(tex_bone_type       );
    }

    hd_cluster_data      .device_array().bind_tex( tex_bulge_strength  );
    hd_offset            .device_array().bind_tex( tex_offset          );
    hd_blending_list     .device_array().bind_tex( tex_blending_list   );
    hd_grid              .device_array().bind_tex( tex_grid            );
    hd_grid_blending_list.device_array().bind_tex( tex_grid_list       );
    hd_grid_bbox         .device_array().bind_tex( tex_grid_bbox       );
}

// -----------------------------------------------------------------------------

void unbind()
{
    binded = false;
    CUDA_SAFE_CALL( hipUnbindTexture(&tex_bone_hrbf)        );
    CUDA_SAFE_CALL( hipUnbindTexture(&tex_bone_precomputed) );

    CUDA_SAFE_CALL( hipUnbindTexture(&tex_blending_list)    );
    CUDA_SAFE_CALL( hipUnbindTexture(&tex_bone_type)        );
    CUDA_SAFE_CALL( hipUnbindTexture(&tex_bulge_strength)   );
    CUDA_SAFE_CALL( hipUnbindTexture(&tex_offset)           );
    CUDA_SAFE_CALL( hipUnbindTexture(&tex_grid)             );
    CUDA_SAFE_CALL( hipUnbindTexture(&tex_grid_list)        );
    CUDA_SAFE_CALL( hipUnbindTexture(&tex_grid_bbox)        );
}

// -----------------------------------------------------------------------------

// =============================================================================

/// @param sid : skeleton identifier
/// @param cell_id : linear index of the 3d cell we want to extract the blending
/// list
/// @param blist : described the sub-skeleton in the
void cell_to_blending_list(Skel_id sid,
                           int cell_id,
                           std::vector<Cluster>& blist)
{
    // Note: if in each cell the list of bones is order from root to leaf
    // then blending list will be also ordered root to leaf
    const Grid*    g    = h_envs[sid]->h_grid;
    const Tree_cu* tree = h_envs[sid]->h_tree_cu_instance;
    const std::list<Bone::Id> &bones_in_cell = g->_grid_cells[cell_id];

    std::vector<bool> cluster_done(tree->_clusters.size(), false);

    for(Bone::Id bone_id: bones_in_cell)
    {
        DBone_id dbone = tree->hidx_to_didx(bone_id);

        Cluster_id clus_id = tree->bone_to_cluster( dbone );
        if( cluster_done[clus_id.id()] ) continue;

        tree->add_cluster( clus_id, blist );

        cluster_done[clus_id.id()] = true;
    }
}

// -----------------------------------------------------------------------------

/// Fill device array : hd_grid_blending_list; hd_offset (only grid_data field);
/// hd_grid; hd_grid_data
static void update_device_grid()
{
#if 1
    assert( !binded );
    hd_grid.fill( -1 );

    // Look up every grids and compute the cells blending list.
    // update offset to access blending list as well
    int offset = 0;
    int grid_offset = 0;
    int off_bone = 0;

    for(unsigned grid_id = 0; grid_id < h_envs.size(); ++grid_id)
    {
        if(h_envs[grid_id] == NULL)
            continue;

        const Grid* grid = h_envs[grid_id]->h_grid;
        const Tree_cu *tree = h_envs[grid_id]->h_tree_cu_instance;

//        grid->build_grid(); should be already done
//        ((Grid *)grid)->build_grid(); // (but isn't always)

        // Get the blending list for each cell, and the total number of resulting clusters.
        std::map<int, std::vector<Cluster> > blist_per_cell;
        int total_size = 0;
        for(int cell_idx: grid->_filled_cells) {
            cell_to_blending_list(grid_id, cell_idx, blist_per_cell[cell_idx]);
            total_size += blist_per_cell[cell_idx].size();
        }

        // Allocate space for these clusters.
        hd_grid_blending_list.realloc(offset + total_size);
        hd_grid_data.realloc(offset + total_size);

        for(int cell_idx: grid->_filled_cells)
        {
            std::vector<Cluster> &blist = blist_per_cell.at(cell_idx);
            if(blist.size() > 0)
                blist[0].datas._blend_type = (EJoint::Joint_t)(tree->_blending_list.size()/2);

            hd_grid[grid_offset + cell_idx] = offset;

            for(const Cluster &c: blist)
            {
                // Convert cluster to cluster_cu and offset bones id to match the concateneted representation
                Cluster_cu clus(c);

                clus.first_bone += off_bone;

                hd_grid_blending_list[offset] = clus;
                hd_grid_data         [offset]._bulge_strength = c.datas._bulge_strength;
                offset++;
            }
        }

        hd_offset[grid_id].grid_data = grid_offset;
        const int res = grid->res();
        grid_offset += res*res*res;

        // Update grid bbox and resolution
        BBox_cu bb = grid->bbox();
        hd_grid_bbox[grid_id*2 + 0] = bb.pmin.to_float4();
        hd_grid_bbox[grid_id*2 + 1] = bb.pmax.to_float4();
        hd_grid_bbox[grid_id*2 + 0].w = (float)res;

        off_bone += h_envs[grid_id]->h_tree_cu_instance->_bone_aranged.size();
    }

    hd_offset.update_device_mem(); // This is also done in update_device_tree maybe we can factorize
    hd_grid.update_device_mem();
    hd_grid_blending_list.update_device_mem();
    hd_grid_data.update_device_mem();
    hd_grid_bbox.update_device_mem();
#endif
}

// -----------------------------------------------------------------------------

/// Fill device array : hd_bone_types; hd_bone_hrbf;
/// hd_bone_precomputed; hd_bulge_strength;
static void fill_separated_bone_types(const std::vector<const Bone*>& generic_bones)
{
    assert( !binded   );
    assert( allocated );

    const int nb_bones = generic_bones.size();
    hd_bone_arrays->resize( nb_bones );

    // For each bone, store the type, and the bone's HRBF and primitive ID.  We can store
    // the IDs even if the bone is in a different mode.
    for(int i = 0; i < nb_bones; i++)
    {
        const Bone* b = generic_bones[i];
        hd_bone_arrays->hd_bone_hrbf[i] = b->get_hrbf();
        hd_bone_arrays->hd_bone_precomputed[i] = b->get_primitive();
        hd_bone_arrays->hd_bone_types[i] = b->get_type();
    }
    // Upload every arrays to GPU
    hd_bone_arrays->update_device_mem();
}

// -----------------------------------------------------------------------------


/// Fill device array : hd_blending_list; hd_offset (only list_data field);
/// h_generic_bones; _hidx_to_didx; _didx_to_hidx;
static void update_device_tree(std::vector<const Bone*> &h_generic_bones)
{
    assert( !binded );
    // Convert host layout to the GPU friendly layout
    // And compute some array sizes.

    int s_blend_list = 0; // Total size of all blending lists
    for(unsigned i = 0; i < h_envs.size(); ++i)
    {
        if(h_envs[i] == NULL)
            continue;

        // Convert tree to GPU layout
        delete h_envs[i]->h_tree_cu_instance;
        h_envs[i]->h_tree_cu_instance = new Tree_cu( h_envs[i]->h_tree );
        s_blend_list += h_envs[i]->h_tree_cu_instance->_blending_list.size();
    }

    // Now we can allocate memory
    hd_offset.malloc( h_envs.size() );

    hd_blending_list.malloc( s_blend_list );
    hd_cluster_data. malloc( s_blend_list );

    _hidx_to_didx.clear();
    _didx_to_hidx.clear();

    // Concatenate bones and blending list.
    // Note that the bone identifiers in the new blending list must
    // be changed to match the list of concatenated bones

    int off_bone  = 0; // Offset to store bones in h_bone_device
    int off_blist = 0; // Offset to store blending list in
    for(unsigned t = 0; t < h_envs.size(); ++t)
    {
        if(h_envs[t] == NULL)
            continue;

        const Tree_cu* tree_cu = h_envs[t]->h_tree_cu_instance;

        for(unsigned i = 0; i < tree_cu->_bone_aranged.size(); ++i){
            DBone_id new_didx = DBone_id(i) + off_bone;
            Hbone_id hidx(t, tree_cu->get_id_bone_aranged( i ) );
            h_generic_bones.push_back(tree_cu->_bone_aranged[i]);
            // Build correspondance between device/host index for the
            // concatenated bones
            _hidx_to_didx[ hidx     ] = new_didx;
            _didx_to_hidx[ new_didx ] = hidx;
        }

        // Concatenate blending list and update bone index accordingly
        std::vector<Cluster>::const_iterator it = tree_cu->_blending_list.begin();
        for(int i = 0; it != tree_cu->_blending_list.end(); ++it, ++i)
        {
            Cluster c = *it;
            c.first_bone += off_bone;
            // Convert in device representation
            Cluster_cu new_c( c );
            hd_blending_list[off_blist + i] = new_c;
            hd_cluster_data [off_blist + i]._bulge_strength = c.datas._bulge_strength;
        }
        // We store nb_pairs in the first element of the list
        assert(tree_cu->_blending_list.size() > 0); // unless we have no elements
        hd_blending_list[off_blist].nb_pairs      = tree_cu->_blending_list.size()/2;

        hd_offset[t].list_data = off_blist;

        off_blist += tree_cu->_blending_list.size();
        off_bone  += tree_cu->_bone_aranged.size();
    }

    // Upload to GPU
    hd_offset.update_device_mem();
    hd_blending_list.update_device_mem();
    hd_cluster_data. update_device_mem();
    assert( off_blist == s_blend_list );
}

// -----------------------------------------------------------------------------

/// Convert CPU representation to GPU
void update_device()
{
    unbind();
    
    // List of concatened bones for all skeletons in 'h_envs'.  Note that a bone may
    // appear in h_generic_bones more than once, if it's used in multiple skeletons.
    std::vector<const Bone*> h_generic_bones;
    update_device_tree(h_generic_bones);

    fill_separated_bone_types( h_generic_bones );
    update_device_grid();
    bind();
}

// -----------------------------------------------------------------------------

void clean_env()
{
    unbind();
    for(unsigned i = 0; i < h_envs.size(); ++i){
        delete h_envs[i];
    }

    h_envs.clear();
    _didx_to_hidx.clear();
    _hidx_to_didx.clear();
    hd_offset.erase();
    hd_offset.update_device_mem();
    hd_grid_blending_list.erase();
    hd_grid_blending_list.update_device_mem();
    hd_grid_data.erase();
    hd_grid_data.update_device_mem();
    hd_grid.erase();
    hd_grid.update_device_mem();
    hd_grid_bbox.erase();
    hd_grid_bbox.update_device_mem();
    hd_blending_list.erase();
    hd_blending_list.update_device_mem();
    hd_bone_arrays->clear();
    hd_bone_arrays->update_device_mem();
    delete hd_bone_arrays;
    hd_bone_arrays = 0;
    allocated = false;
}

// -----------------------------------------------------------------------------

void alloc_hd_grid()
{
    assert( binded );
    unbind();

    int total_size = 0;
    for(unsigned i = 0; i < h_envs.size(); ++i){
        if(h_envs[i] == NULL)
            continue;
        const int res = h_envs[i]->h_grid->res();
        total_size += res*res*res;
    }
    hd_grid.malloc(total_size, -1);
    hd_grid_bbox.malloc( h_envs.size() * 2 ); // Two points for a bbox

    bind();
}

// -----------------------------------------------------------------------------

void init_env()
{
    if( !allocated)
    {
        hd_bone_arrays = new Bone_tex();
        allocated = true;
        bind();
    }
}

// -----------------------------------------------------------------------------

Skel_id new_skel_instance(const std::vector<const Bone*>& bones,
                          const std::map<Bone::Id, Bone::Id>& parents,
                          int grid_res)
{
    SkeletonEnv *env = new SkeletonEnv();
    env->h_tree = new Tree(bones, parents);
    env->h_grid = new Grid(env->h_tree, grid_res);
    env->h_grid->build_grid();

    // Find an empty slot.
    int id;
    for(id = 0; id < (int) h_envs.size(); ++id)
    {
        if(h_envs[id] == NULL)
            break;
    }

    // Add a slot if needed.
    if(id >= h_envs.size())
        h_envs.push_back(NULL);
    
    h_envs[id] = env;

    alloc_hd_grid();
    update_device();
    return id;
}

// -----------------------------------------------------------------------------

void delete_skel_instance(Skel_id skel_id)
{
    assert(skel_id < h_envs.size());
    assert(skel_id >= 0);

    // Set the slot to NULL to allow reuse.
    delete h_envs[skel_id];
    h_envs[skel_id] = NULL;
    
    alloc_hd_grid();
    update_device();
}

// -----------------------------------------------------------------------------

void update_bones_data(Skel_id i)
{
    h_envs[i]->h_grid->build_grid();
    update_device();
}

// -----------------------------------------------------------------------------

void update_joints_data(Skel_id i, const std::map<Bone::Id, Joint_data>& joints)
{
    h_envs[i]->h_tree->set_joints_data( joints );
    h_envs[i]->h_grid->build_grid();
    update_device();
}

// -----------------------------------------------------------------------------

void set_grid_res(Skel_id i, int res)
{
    assert( res > 0);
    h_envs[i]->h_grid->set_res( res );
    alloc_hd_grid();
    update_device();
}

// -----------------------------------------------------------------------------

DBone_id bone_hidx_to_didx(Skel_id skel_id, Bone::Id bone_hidx)
{
    // TODO: array of maps by skeleton ids would be more efficient
    Hbone_id hidx(skel_id, bone_hidx);
    return Std_utils::find( _hidx_to_didx, hidx);
}

// -----------------------------------------------------------------------------

Bone::Id bone_didx_to_hidx(Skel_id skel_id, DBone_id bone_didx)
{
    Hbone_id hid = Std_utils::find( _didx_to_hidx, bone_didx);
    assert( hid._skel_id == skel_id);
    return hid._bone_id;
}
}// End Skeleton_env ===========================================================
