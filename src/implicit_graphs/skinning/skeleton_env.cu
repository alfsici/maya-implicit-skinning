#include "hip/hip_runtime.h"
/*
 Implicit skinning
 Copyright (C) 2013 Rodolphe Vaillant, Loic Barthe, Florian Cannezin,
 Gael Guennebaud, Marie Paule Cani, Damien Rohmer, Brian Wyvill,
 Olivier Gourmel

 This program is free software: you can redistribute it and/or modify
 it under the terms of the GNU General Public License 3 as published by
 the Free Software Foundation.

 This program is distributed in the hope that it will be useful,
 but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 GNU General Public License for more details.

 You should have received a copy of the GNU General Public License
 along with this program. If not, see <http://www.gnu.org/licenses/>
 */
#include "skeleton_env.hpp"
#include "skeleton.hpp"

#include "std_utils.hpp"
#include "grid.hpp"
#include "tree_cu.hpp"
#include "tree.hpp"
#include <list>
#include <deque>
#include <map>
#include <set>

// =============================================================================
namespace Skeleton_env {
// =============================================================================

bool allocated = false;

// from file skeleton_env_tex_binded.hpp
extern bool binded;

// -----------------------------------------------------------------------------
/// @name GPU friendly datas
// -----------------------------------------------------------------------------

Bone_tex* hd_bone_arrays = 0;

/// Concatenated blendind list for every skeletons
Cuda_utils::HD_Array<Cluster_cu> hd_blending_list;

Cuda_utils::HD_Array<Cluster_data> hd_cluster_data;

/// Concatenated blendind list for every skeletons in each grid cell not empty.
/// Each cell store a single blending list.
/**
 * @code
 *  |-list0-|-list1-|-list2-| |-list0-|-list1-|
 *  *-----------------------* *---------------*
 *            skel0                 skel1
 * @endcode
*/
Cuda_utils::HD_Array<Cluster_cu> hd_grid_blending_list;

/// Concatenated datas corresponding to clusters listed hd_grid_blending_list
Cuda_utils::HD_Array<Cluster_data> hd_grid_data;
/// Table of indirection which maps grid cells to blending list.
/// hd_grid[ hd_offset[Skel_id].grid_data + cell_idx] == offset in hd_grid_blending_list or -1 if empty cell
Cuda_utils::HD_Array<int> hd_grid;

Cuda_utils::HD_Array<float4> hd_grid_bbox;

/// d_offset[Skel_id] == offset in lists or grid
Cuda_utils::HD_Array<Offset>  hd_offset; // maybe we should cut in half offset and allocate it in advance at each new skeleton instances

// -----------------------------------------------------------------------------
/// @name CPU friendly datas
// -----------------------------------------------------------------------------

/// List of concatened bones for every skeletons in 'h_tree_instances'
std::vector<Bone*> h_generic_bones;
/// user idx to device bone idx
std::map<Hbone_id, DBone_id> _hidx_to_didx;
/// device bone idx to user idx
std::map<DBone_id, Hbone_id> _didx_to_hidx;


/// List of skeletons instances, a skeleton must be a tree with one connex
/// componant.
std::deque<Tree*>    h_tree_instances;
/// Prepared tree data to go on GPU
std::deque<Tree_cu*> h_tree_cu_instances;

/// Acceleration structure for the different skeletons in the environment
std::deque<Grid*> h_grid;



// =============================================================================

/// @param sid : skeleton identifier
/// @param cell_id : linear index of the 3d cell we want to extract the blending
/// list
/// @param blist : described the sub-skeleton in the
void cell_to_blending_list(Skel_id sid,
                           int cell_id,
                           std::list<Cluster>& blist)
{
    // Note: if in each cell the list of bones is order from root to leaf
    // then blending list will be also ordered root to leaf
    const Grid*    g    = h_grid[sid];
    const Tree_cu* tree = h_tree_cu_instances[sid];
    const std::list<Bone::Id>& bones_in_cell = g->_grid_cells[cell_id];

    std::vector<bool> cluster_done;
    cluster_done.resize(tree->_clusters.size(), false);

    Tree_cu::BList blending_list( *(h_tree_cu_instances[sid]) );
    std::list<Bone::Id>::const_iterator bones_it = bones_in_cell.begin();
    for(; bones_it != bones_in_cell.end(); ++bones_it)
    {
        DBone_id dbone = bone_hidx_to_didx(sid, *bones_it);

        Cluster_id clus_id = tree->bone_to_cluster( dbone );
        if( cluster_done[clus_id.id()] ) continue;

        blending_list.add_cluster( clus_id );

        cluster_done[clus_id.id()] = true;
    }

    // HACK: this should be done outside this function when convberting cluster
    // to cluster_cu
    if(blending_list._list.size() > 0){
        blending_list._list.begin()->datas._ctrl_id    = blending_list._nb_singletons;
        blending_list._list.begin()->datas._blend_type = (EJoint::Joint_t)blending_list._nb_pairs;
    }

    blist.swap( blending_list._list );
}

// -----------------------------------------------------------------------------

/// Fill device array : hd_grid_blending_list; hd_offset (only grid_data field);
/// hd_grid; hd_grid_data
static void update_device_grid()
{
#if 1
    assert( !binded );
    hd_grid.fill( -1 );

    // Look up every grids and compute the cells blending list.
    // update offset to access blending list as well
    std::deque< std::list<Cluster>* > cells_blist; // TODO/ <- pre-allocate this in alloc_hd_grid()
    int offset = 0;
    int grid_offset = 0;
    for(unsigned grid_id = 0; grid_id < h_grid.size(); ++grid_id)
    {
        Grid* grid = h_grid[grid_id];
//        grid->build_grid(); should be already done

        std::set<int>::const_iterator it = grid->_filled_cells.begin();
        for( ; it != grid->_filled_cells.end(); ++it)
        {
            int cell_idx = *it;

            std::list<Cluster>* blist = new std::list<Cluster>();
            cells_blist.push_back( blist );
            cell_to_blending_list(grid_id, cell_idx, *blist );

            hd_grid[grid_offset + cell_idx] = offset;
            offset += (int)blist->size();
        }

        hd_offset[grid_id].grid_data = grid_offset;
        const int res = grid->res();
        grid_offset += res*res*res;

        // Update grid bbox and resolution
        BBox_cu bb = grid->bbox();
        hd_grid_bbox[grid_id*2 + 0] = bb.pmin.to_float4();
        hd_grid_bbox[grid_id*2 + 1] = bb.pmax.to_float4();
        hd_grid_bbox[grid_id*2 + 0].w = (float)res;
    }

    hd_grid_blending_list.malloc( offset );
    hd_grid_data.malloc( offset );

    // Concatenate blending lists compute new concatenated bone index:
    int grid_id = 0;
    int off_bone = 0;
    int blist_offset = 0;
    for(unsigned i = 0; i < cells_blist.size(); ++i)
    {
        const std::list<Cluster>& blist = *(cells_blist[i]);
        std::list<Cluster>::const_iterator it = blist.begin();
        for(unsigned l = 0; it != blist.end(); ++it, ++l)
        {

            // Convert cluster to cluster_cu and offset bones id to match the concateneted representation
            Cluster c = *it;
            Cluster_cu clus(c);

            clus.first_bone += off_bone;

            hd_grid_blending_list[blist_offset + l] = clus;
            hd_grid_data         [blist_offset + l]._bulge_strength = c.datas._bulge_strength;;
        }

        blist_offset += blist.size();
        delete cells_blist[i];
        cells_blist[i] = 0;

        // Compute grid identifier and bone offset
        int res = h_grid[grid_id]->res();
        if( (hd_offset[grid_id].grid_data +  res*res*res) < ((int)i+1)){
            grid_id++;
            if( grid_id < (int)h_grid.size() )
                off_bone += h_tree_cu_instances[grid_id]->_bone_aranged.size();
        }
    }

    hd_offset.update_device_mem(); // This is also done in update_device_tree maybe we can factorize
    hd_grid.update_device_mem();
    hd_grid_blending_list.update_device_mem();
    hd_grid_data.update_device_mem();
    hd_grid_bbox.update_device_mem();
#endif
}

// -----------------------------------------------------------------------------

/// Fill device array : hd_bone_types; hd_bone; hd_bone_hrbf; hd_bone_cylinder;
/// hd_bone_precomputed; hd_bulge_strength;
static void fill_separated_bone_types(const std::vector<Bone*>& generic_bones)
{
    assert( !binded   );
    assert( allocated );

    const int nb_bones = generic_bones.size();
    hd_bone_arrays->resize( nb_bones );
    for(int i = 0; i < nb_bones; i++)
    {
        const Bone* b = generic_bones[i];
        const int btype = b->get_type();
        switch( btype )
        {
        case EBone::CYLINDER:
            hd_bone_arrays->hd_bone_cylinder[i] = ((Bone_cylinder*)b)->get_cylinder();
            break;
        case EBone::HRBF:
            hd_bone_arrays->hd_bone_hrbf[i] = ((Bone_hrbf*)b)->get_hrbf();
            break;
        case EBone::SSD:
            /* There is no data to store for ssd bones*/
            break;
        case EBone::PRECOMPUTED:
            hd_bone_arrays->hd_bone_precomputed[i] = ((Bone_precomputed*)b)->get_primitive();
            break;
        }

        hd_bone_arrays->hd_bone      [i] = b->get_bone_cu();
        hd_bone_arrays->hd_bone_types[i] = btype;
    }
    // Upload every arrays to GPU
    hd_bone_arrays->update_device_mem();
}

// -----------------------------------------------------------------------------


/// Fill device array : hd_blending_list; hd_offset (only list_data field);
/// h_generic_bones; _hidx_to_didx; _didx_to_hidx;
static void update_device_tree()
{
    assert( !binded );
    // Convert host layout to the GPU friendly layout
    // And compute some array sizes.
    std::deque<Tree_cu*>& tree_cu_list = h_tree_cu_instances;

    int nb_bones_all = 0; // Number of bones for every concatenated skels
    int s_blend_list = 0; // Size of the concatenated blending list
    for(unsigned i = 0; i < h_tree_instances.size(); ++i)
    {
        // Convert tree to GPU layout
        delete tree_cu_list[i];
        tree_cu_list[i] = new Tree_cu( h_tree_instances[i] );
        nb_bones_all += h_tree_instances[i]->bone_size();
        s_blend_list += tree_cu_list[i]->_blending_list._list.size();
    }

    // Now we can allocate memory
    hd_offset.malloc( h_tree_instances.size() );
    h_generic_bones.resize( nb_bones_all );

    hd_blending_list.malloc( s_blend_list );
    hd_cluster_data. malloc( s_blend_list );

    _hidx_to_didx.clear();
    _didx_to_hidx.clear();

    // Concatenate bones and blending list.
    // Note that the bone identifiers in the new blending list must
    // be changed to match the list of concatenated bones

    int off_bone  = 0; // Offset to store bones in h_bone_device
    int off_blist = 0; // Offset to store blending list in
    for(unsigned t = 0; t < h_tree_instances.size(); ++t)
    {
        const Tree_cu* tree_cu = tree_cu_list[t];

        for(unsigned i = 0; i < tree_cu->_bone_aranged.size(); ++i){
            DBone_id new_didx = DBone_id(i) + off_bone;
            Hbone_id hidx(t, tree_cu->get_id_bone_aranged( i ) );
            h_generic_bones[new_didx.id()] = tree_cu->_bone_aranged[i];
            // Build correspondance between device/host index for the
            // concatenated bones
            _hidx_to_didx[ hidx     ] = new_didx;
            _didx_to_hidx[ new_didx ] = hidx;
        }

        // Concatenate blending list and update bone index accordingly
        std::list<Cluster>::const_iterator it = tree_cu->_blending_list._list.begin();
        for(int i = 0; it != tree_cu->_blending_list._list.end(); ++it, ++i)
        {
            Cluster c = *it;
            c.first_bone += off_bone;
            // Convert in device representation
            Cluster_cu new_c( c );
            hd_blending_list[off_blist + i] = new_c;
            hd_cluster_data [off_blist + i]._bulge_strength = c.datas._bulge_strength;
        }
        // We store nb_pairs and nb_singletons in the first element of the list
        hd_blending_list[off_blist].nb_pairs      = tree_cu->_blending_list._nb_pairs;
        hd_blending_list[off_blist].nb_singletons = tree_cu->_blending_list._nb_singletons;

        hd_offset[t].list_data = off_blist;

        off_blist += tree_cu->_blending_list._list.size();
        off_bone  += tree_cu->_bone_aranged.size();
    }

    // Upload to GPU
    hd_offset.update_device_mem();
    hd_blending_list.update_device_mem();
    hd_cluster_data. update_device_mem();
    assert( off_bone  == nb_bones_all );
    assert( off_blist == s_blend_list );
}

// -----------------------------------------------------------------------------

/// Convert CPU representation to GPU
void update_device()
{
    unbind();
    update_device_tree();
    fill_separated_bone_types( h_generic_bones );
    update_device_grid();
    bind();
}

// -----------------------------------------------------------------------------

void clean_env()
{
    unbind();
    for(unsigned i = 0; i < h_tree_instances.size(); ++i){
        delete h_tree_instances[i];
        delete h_tree_cu_instances[i];
        delete h_grid[i];
    }

    h_tree_instances.clear();
    h_tree_cu_instances.clear();
    h_grid.clear();
    _didx_to_hidx.clear();
    _hidx_to_didx.clear();
    h_generic_bones.clear();
    hd_offset.erase();
    hd_offset.update_device_mem();
    hd_grid_blending_list.erase();
    hd_grid_blending_list.update_device_mem();
    hd_grid_data.erase();
    hd_grid_data.update_device_mem();
    hd_grid.erase();
    hd_grid.update_device_mem();
    hd_grid_bbox.erase();
    hd_grid_bbox.update_device_mem();
    hd_blending_list.erase();
    hd_blending_list.update_device_mem();
    hd_bone_arrays->clear();
    hd_bone_arrays->update_device_mem();
    delete hd_bone_arrays;
    hd_bone_arrays = 0;
    allocated = false;
}

// -----------------------------------------------------------------------------

void alloc_hd_grid()
{
    assert( binded );
    unbind();

    int total_size = 0;
    for(unsigned i = 0; i < h_grid.size(); ++i){
        const int res = h_grid[i]->res();
        total_size += res*res*res;
    }
    hd_grid.malloc(total_size, -1);
    hd_grid_bbox.malloc( h_grid.size() * 2 ); // Two points for a bbox

    bind();
}

// -----------------------------------------------------------------------------

void init_env()
{
    if( !allocated)
    {
        hd_bone_arrays = new Bone_tex();
        allocated = true;
        bind();
    }
}

// -----------------------------------------------------------------------------

Skel_id new_skel_instance(Bone::Id root_idx,
                          const std::vector<Bone*>& bones,
                          const std::vector<int>& parents)
{
    int id = (unsigned)h_tree_instances.size();

    Tree* tree = new Tree(root_idx, bones, parents);
    h_tree_instances.push_back( tree );
    h_tree_cu_instances.push_back( 0 );
    h_grid.push_back( new Grid( tree ) );
    alloc_hd_grid();
    update_device();
    return id;
}

// -----------------------------------------------------------------------------

void delete_skel_instance(Skel_id i)
{
    delete h_grid[i];
    delete h_tree_instances[i];
    delete h_tree_cu_instances[i];
    h_tree_instances.erase( h_tree_instances.begin() +i);
    h_tree_cu_instances.erase( h_tree_cu_instances.begin() + i);
    h_grid.erase( h_grid.begin() + i);
    alloc_hd_grid();
    update_device();
}

// -----------------------------------------------------------------------------

void update_bones_data(Skel_id i, const std::vector<Bone*>& bones)
{
    h_tree_instances[i]->set_bones( bones );
    h_grid[i]->build_grid();
    update_device();
}

// -----------------------------------------------------------------------------

void update_joints_data(Skel_id i, const std::vector<Joint_data>& joints)
{
    h_tree_instances[i]->set_joints_data( joints );
    h_grid[i]->build_grid();
    update_device();
}

// -----------------------------------------------------------------------------

void set_grid_res(Skel_id i, int res)
{
    assert( res > 0);
    h_grid[i]->set_res( res );
    alloc_hd_grid();
    update_device();
}

// -----------------------------------------------------------------------------

DBone_id bone_hidx_to_didx(Skel_id skel_id, Bone::Id bone_hidx)
{
    // TODO: array of maps by skeleton ids would be more efficient
    Hbone_id hidx(skel_id, bone_hidx);
    return Std_utils::find( _hidx_to_didx, hidx);
}

// -----------------------------------------------------------------------------

Bone::Id bone_didx_to_hidx(Skel_id skel_id, DBone_id bone_didx)
{
    Hbone_id hid = Std_utils::find( _didx_to_hidx, bone_didx);
    assert( hid._skel_id == skel_id);
    return hid._bone_id;
}
}// End Skeleton_env ===========================================================

// -----------------------------------------------------------------------------
/// Debugging
// -----------------------------------------------------------------------------

#include "blending_env_tex.hpp"
#include "skeleton_env_tex.hpp"

#include "vbo_primitives.hpp"
#include "glsave.hpp"
#include"color.hpp"

// Some externs from "globals.hpp" ---------------------------------------------
extern VBO_primitives g_primitive_printer;
extern Prim_id g_cube_vbo;
// End externs from "globals.hpp" ----------------------------------------------

// =============================================================================
namespace Skeleton_env {
// =============================================================================

/// Draw a grid cell (red if empty or green)
void draw_cell(const Point_cu& org, Vec3_cu lengths, bool empty)
{
    glPushMatrix();

    glTranslatef(org.x, org.y, org.z);
    glScalef(lengths.x, lengths.y, lengths.z);

    if( empty ) glColor3f(1.f, 0.f, 0.f);
    else        glColor3f(0.f, 1.f, 0.f);
    g_primitive_printer.draw( g_cube_vbo );
    glPopMatrix();
}



// -----------------------------------------------------------------------------

/// Evaluate a point given a skeleton's grid
void draw_sample(Skel_id i, Vec3_cu pos)
{
    Cluster_id clus_off = fetch_grid_blending_list_offset(i, pos);

    if( clus_off.is_valid() ){
        Cluster_cu clus = fetch_grid_blending_list( clus_off );
        // In the first cluster we don't store the blending type and controller id
        const int nb_pairs      = clus.nb_pairs;
        const int nb_singletons = clus.nb_singletons;

//        if(nb_pairs == 0 && nb_singletons == 0) glColor3f(0.f, 0.f, 0.f);
//        else                                    glColor3f(0.f, 1.f, 0.f);

        float c = (float)(nb_pairs*2 + nb_singletons) / (float)(h_tree_instances[i]->bone_size());
        Color::heat_color(1.f - c).set_gl_state();

        /*
        int cid = 0;
        for(; cid < nb_pairs; ++cid) {
            Cluster_cu clus0 = fetch_grid_blending_list( clus_off + cid*2 + 0);
            Cluster_cu clus1 = fetch_grid_blending_list( clus_off + cid*2 + 0);
            int a = 0;
            a++;
        }

        cid *= 2;
        int off = cid;
        for(; cid < (nb_singletons + off); ++cid){
            Cluster_cu clus0 = fetch_grid_blending_list( clus_off + cid);
            int a = 0;
            a++;
        }
        */
    }
    else
        glColor3f(1.f, 0.f, 0.f);

    glBegin(GL_POINTS);
    glVertex3f(pos.x, pos.y, pos.z);
    glEnd();
}

// -----------------------------------------------------------------------------

void draw_grids()
{
    GLPolygonModeSave poly_mode( GL_LINE );
    GLEnabledSave light(GL_LIGHTING, true, false);
    GLEnabledSave depth(GL_DEPTH_TEST, true, true);
    GLPointSizeSave size(5.f);
    for(unsigned gid = 0; gid < h_grid.size(); ++gid)
    {
        BBox_cu bb;
        int res;

        float4 f0 = hd_grid_bbox[gid*2 + 0];
        float4 f1 = hd_grid_bbox[gid*2 + 1];

        bb = BBox_cu( Point_cu(f0.x, f0.y, f0.z),
                      Point_cu(f1.x, f1.y, f1.z) );
        res = (int)f0.w;

        Vec3i_cu size_grid(res, res, res);
        Vec3_cu cell_lengths = bb.lengths().div( (Vec3_cu)size_grid );
        for(Idx3_cu idx(size_grid, 0); idx.is_in(); ++idx)
        {
            Point_cu pt = bb.pmin + cell_lengths.mult( (Vec3_cu)idx.to_3d() );
//            int offset = hd_offset[gid].grid_data;
//            int state = hd_grid[ offset + idx.to_linear() ];
//            draw_cell(pt, cell_lengths, state < 0);

//            const int cpt = 10;
//            for (int i = 0; i < cpt; ++i)
                draw_sample(gid, pt + (cell_lengths / (float)2.f) /** i*/);

        }

    }
}

}// End Skeleton_env ===========================================================
